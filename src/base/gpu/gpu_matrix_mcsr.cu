#include "hip/hip_runtime.h"
// **************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2015  PARALUTION Labs UG (haftungsbeschränkt) & Co. KG
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRA 706051
//                        Vertreten durch:
//                        PARALUTION Labs Verwaltungs UG (haftungsbeschränkt)
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRB 721277
//                        Geschäftsführer: Dimitar Lukarski, Nico Trost
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// **************************************************************************



// PARALUTION version 1.0.0 


#include "../../utils/def.hpp"
#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_mcsr.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_mcsr.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_mcsr.hpp"
#include "gpu_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <hip/hip_runtime.h>
#include <hipsparse.h>

namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixMCSR<ValueType>::GPUAcceleratorMatrixMCSR() {

  // no default constructors
  LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixMCSR<ValueType>::GPUAcceleratorMatrixMCSR(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "GPUAcceleratorMatrixMCSR::GPUAcceleratorMatrixMCSR()",
            "constructor with local_backend");

  this->mat_.row_offset = NULL;  
  this->mat_.col = NULL;  
  this->mat_.val = NULL;
  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

}


template <typename ValueType>
GPUAcceleratorMatrixMCSR<ValueType>::~GPUAcceleratorMatrixMCSR() {

  LOG_DEBUG(this, "GPUAcceleratorMatrixMCSR::~GPUAcceleratorMatrixMCSR()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixMCSR<ValueType>");

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::AllocateMCSR(const int nnz, const int nrow, const int ncol) {

  assert(nnz >= 0);
  assert(ncol >= 0);
  assert(nrow >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nnz > 0) {

    allocate_gpu(nrow+1, &this->mat_.row_offset);
    allocate_gpu(nnz,    &this->mat_.col);
    allocate_gpu(nnz,    &this->mat_.val);
    
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nrow+1, mat_.row_offset);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, mat_.col);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, mat_.val);

    this->nrow_ = nrow;
    this->ncol_ = ncol;
    this->nnz_  = nnz;

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    free_gpu(&this->mat_.row_offset);
    free_gpu(&this->mat_.col);
    free_gpu(&this->mat_.val);

    this->nrow_ = 0;
    this->ncol_ = 0;
    this->nnz_  = 0;

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixMCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixMCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateMCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    hipMemcpy(this->mat_.row_offset,     // dst
               cast_mat->mat_.row_offset, // src
               (this->get_nrow()+1)*sizeof(int), // size
               hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(this->mat_.col,     // dst
               cast_mat->mat_.col, // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(this->mat_.val,     // dst
               cast_mat->mat_.val, // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyHostToDevice);    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixMCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixMCSR<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateMCSR(this->get_nnz(), this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    hipMemcpy(cast_mat->mat_.row_offset, // dst
               this->mat_.row_offset,     // src
               (this->get_nrow()+1)*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(cast_mat->mat_.col, // dst
               this->mat_.col,     // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(cast_mat->mat_.val, // dst
               this->mat_.val,     // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyDeviceToHost);    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     
   
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixMCSR<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixMCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateMCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    hipMemcpy(this->mat_.row_offset,         // dst
               gpu_cast_mat->mat_.row_offset, // src
               (this->get_nrow()+1)*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(this->mat_.col,         // dst
               gpu_cast_mat->mat_.col, // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(this->mat_.val,         // dst
               gpu_cast_mat->mat_.val, // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    
  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixMCSR<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixMCSR<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateMCSR(dst->get_nnz(), dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    hipMemcpy(gpu_cast_mat->mat_.row_offset, // dst
               this->mat_.row_offset,         // src
               (this->get_nrow()+1)*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(gpu_cast_mat->mat_.col, // dst
               this->mat_.col,         // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(gpu_cast_mat->mat_.val, // dst
               this->mat_.val,         // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyDeviceToHost);    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     
   
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::CopyFromHostAsync(const HostMatrix<ValueType> &src) {

  const HostMatrixMCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixMCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateMCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    hipMemcpyAsync(this->mat_.row_offset,     // dst
                    cast_mat->mat_.row_offset, // src
                    (this->get_nrow()+1)*sizeof(int), // size
                    hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpyAsync(this->mat_.col,     // dst
                    cast_mat->mat_.col, // src
                    this->get_nnz()*sizeof(int), // size
                    hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpyAsync(this->mat_.val,     // dst
                    cast_mat->mat_.val, // src
                    this->get_nnz()*sizeof(ValueType), // size
                    hipMemcpyHostToDevice);    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::CopyToHostAsync(HostMatrix<ValueType> *dst) const {

  HostMatrixMCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixMCSR<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateMCSR(this->get_nnz(), this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    hipMemcpyAsync(cast_mat->mat_.row_offset, // dst
                    this->mat_.row_offset,     // src
                    (this->get_nrow()+1)*sizeof(int), // size
                    hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpyAsync(cast_mat->mat_.col, // dst
                    this->mat_.col,     // src
                    this->get_nnz()*sizeof(int), // size
                    hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpyAsync(cast_mat->mat_.val, // dst
                    this->mat_.val,     // src
                    this->get_nnz()*sizeof(ValueType), // size
                    hipMemcpyDeviceToHost);    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     
   
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::CopyFromAsync(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixMCSR<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixMCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateMCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    hipMemcpy(this->mat_.row_offset,         // dst
               gpu_cast_mat->mat_.row_offset, // src
               (this->get_nrow()+1)*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(this->mat_.col,         // dst
               gpu_cast_mat->mat_.col, // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(this->mat_.val,         // dst
               gpu_cast_mat->mat_.val, // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    
  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHostAsync(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::CopyToAsync(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixMCSR<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixMCSR<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateMCSR(dst->get_nnz(), dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    hipMemcpy(gpu_cast_mat->mat_.row_offset, // dst
               this->mat_.row_offset,         // src
               (this->get_nrow()+1)*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(gpu_cast_mat->mat_.col, // dst
               this->mat_.col,         // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    hipMemcpy(gpu_cast_mat->mat_.val, // dst
               this->mat_.val,         // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyDeviceToHost);    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);     
   
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHostAsync(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
bool GPUAcceleratorMatrixMCSR<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;

  const GPUAcceleratorMatrixMCSR<ValueType>   *cast_mat_mcsr;
  
  if ((cast_mat_mcsr = dynamic_cast<const GPUAcceleratorMatrixMCSR<ValueType>*> (&mat)) != NULL) {

      this->CopyFrom(*cast_mat_mcsr);
      return true;

  }

  /*
  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {
    
    this->Clear();
    
    FATAL_ERROR(__FILE__, __LINE__);
    
    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = cast_mat_csr->get_nnz();
    
    return true;
    
  }
  */

  return false;

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::Apply(const BaseVector<ValueType> &in, BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in);
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_mcsr_spmv_scalar<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col,
                                                                       this->mat_.val,
                                                                       cast_in->vec_, cast_out->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixMCSR<ValueType>::ApplyAdd(const BaseVector<ValueType> &in, const ValueType scalar,
                                                   BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in);
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_mcsr_add_spmv_scalar<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col,
                                                                           this->mat_.val,
                                                                           scalar, cast_in->vec_, cast_out->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}


template class GPUAcceleratorMatrixMCSR<double>;
template class GPUAcceleratorMatrixMCSR<float>;

}
