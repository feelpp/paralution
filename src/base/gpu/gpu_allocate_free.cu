// **************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2015  PARALUTION Labs UG (haftungsbeschränkt) & Co. KG
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRA 706051
//                        Vertreten durch:
//                        PARALUTION Labs Verwaltungs UG (haftungsbeschränkt)
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRB 721277
//                        Geschäftsführer: Dimitar Lukarski, Nico Trost
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// **************************************************************************



// PARALUTION version 1.0.0 


#include "../../utils/def.hpp"
#include "gpu_allocate_free.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "../../utils/allocate_free.hpp"

#include <cmath>

#include <hip/hip_runtime.h>

namespace paralution {

#ifdef PARALUTION_CUDA_PINNED_MEMORY

template <typename DataType>
void allocate_host(const int size, DataType **ptr) {

  LOG_DEBUG(0, "allocate_host()",
            size);

  if (size > 0) {

    assert(*ptr == NULL);
    
    //    *ptr = new DataType[size];

    hipHostMalloc((void **)ptr, size*sizeof(DataType));
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    LOG_DEBUG(0, "allocate_host()",
              *ptr);
    
    assert(*ptr != NULL);
  }

}

template <typename DataType>
void free_host(DataType **ptr) {

  LOG_DEBUG(0, "free_host()",
            *ptr);

  assert(*ptr != NULL);

  //  delete[] *ptr;
  hipHostFree(*ptr);
  CHECK_CUDA_ERROR(__FILE__, __LINE__);
  
  *ptr = NULL;

}

#endif

template <typename DataType>
void allocate_gpu(const int size, DataType **ptr) {

  LOG_DEBUG(0, "allocate_gpu()",
            size);

  if (size > 0) {

    assert(*ptr == NULL);
    
    hipMalloc( (void **)ptr, size*sizeof(DataType));
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    assert(*ptr != NULL);
  }

}

template <typename DataType>
void free_gpu(DataType **ptr) {

  LOG_DEBUG(0, "free_gpu()",
            *ptr);

  assert(*ptr != NULL);
  
  hipFree(*ptr);
  CHECK_CUDA_ERROR(__FILE__, __LINE__);
  
  *ptr = NULL;

}

template <typename DataType>
void set_to_zero_gpu(const int blocksize,
                     const int max_threads,
                     const int size, DataType *ptr) {

  LOG_DEBUG(0, "set_to_zero_gpu()",
            "size =" << size << 
            " ptr=" << ptr);
  
  if (size > 0) {

    assert(ptr != NULL);

    hipMemset(ptr, 0, size*sizeof(DataType));
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    /*
    int s = size ;
    int k = (size/blocksize)/max_threads + 1;       
    if (k > 1) s = size / k;
    

    dim3 BlockSize(blocksize);
    dim3 GridSize(s / blocksize + 1);

    kernel_set_to_zeros<DataType, int> <<<GridSize, BlockSize>>> (size, ptr);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
    */

    /*
    // 1D accessing, no stride
    dim3 BlockSize(blocksize);
    dim3 GridSize(size / blocksize + 1);

    kernel_set_to_zeros<DataType, int> <<<GridSize, BlockSize>>> (size, ptr);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
    */
    
  }

}

template <typename DataType>
void set_to_one_gpu(const int blocksize,
                    const int max_threads,
                    const int size, DataType *ptr) {

  LOG_DEBUG(0, "set_to_zero_gpu()",
            "size =" << size << 
            " ptr=" << ptr);

  if (size > 0) {

    assert(ptr != NULL);

/*
    int s = size ;
    int k = (size/blocksize)/max_threads + 1;
    if (k > 1) s = size / k;

    dim3 BlockSize(blocksize);
    dim3 GridSize(s / blocksize + 1);

    kernel_set_to_ones<DataType, int> <<<GridSize, BlockSize>>> (size, ptr);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);
*/

    // 1D accessing, no stride
    dim3 BlockSize(blocksize);
    dim3 GridSize(size / blocksize + 1);

    kernel_set_to_ones<DataType, int> <<<GridSize, BlockSize>>> (size, ptr);

  }

}


#ifdef PARALUTION_CUDA_PINNED_MEMORY

template void allocate_host<float                >(const int size, float                **ptr);
template void allocate_host<double               >(const int size, double               **ptr);
template void allocate_host<int                  >(const int size, int                  **ptr);
template void allocate_host<unsigned int         >(const int size, unsigned int         **ptr);
template void allocate_host<char                 >(const int size, char                 **ptr);

template void free_host<float                >(float                **ptr);
template void free_host<double               >(double               **ptr);
template void free_host<int                  >(int                  **ptr);
template void free_host<unsigned int         >(unsigned int         **ptr);
template void free_host<char                 >(char                 **ptr);

#endif

template void allocate_gpu<float                >(const int size, float                **ptr);
template void allocate_gpu<double               >(const int size, double               **ptr);
template void allocate_gpu<int                  >(const int size, int                  **ptr);
template void allocate_gpu<unsigned int         >(const int size, unsigned int         **ptr);
template void allocate_gpu<char                 >(const int size, char                 **ptr);

template void free_gpu<float                >(float                **ptr);
template void free_gpu<double               >(double               **ptr);
template void free_gpu<int                  >(int                  **ptr);
template void free_gpu<unsigned int         >(unsigned int         **ptr);
template void free_gpu<char                 >(char                 **ptr);

template void set_to_zero_gpu<float                >(const int blocksize, const int max_threads, const int size, float                *ptr);
template void set_to_zero_gpu<double               >(const int blocksize, const int max_threads, const int size, double               *ptr);
template void set_to_zero_gpu<int                  >(const int blocksize, const int max_threads, const int size, int                  *ptr);
template void set_to_zero_gpu<unsigned int         >(const int blocksize, const int max_threads, const int size, unsigned int         *ptr);
template void set_to_zero_gpu<char                 >(const int blocksize, const int max_threads, const int size, char                 *ptr);


template void set_to_one_gpu<float                >(const int blocksize, const int max_threads, const int size, float                *ptr);
template void set_to_one_gpu<double               >(const int blocksize, const int max_threads, const int size, double               *ptr);
template void set_to_one_gpu<int                  >(const int blocksize, const int max_threads, const int size, int                  *ptr);
template void set_to_one_gpu<unsigned int         >(const int blocksize, const int max_threads, const int size, unsigned int         *ptr);
template void set_to_one_gpu<char                 >(const int blocksize, const int max_threads, const int size, char                 *ptr);

}
