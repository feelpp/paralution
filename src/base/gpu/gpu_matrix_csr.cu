#include "hip/hip_runtime.h"
// **************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2015  PARALUTION Labs UG (haftungsbeschränkt) & Co. KG
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRA 706051
//                        Vertreten durch:
//                        PARALUTION Labs Verwaltungs UG (haftungsbeschränkt)
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRB 721277
//                        Geschäftsführer: Dimitar Lukarski, Nico Trost
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// **************************************************************************



// PARALUTION version 1.0.0 


#include "../../utils/def.hpp"
#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_coo.hpp"
#include "gpu_matrix_dia.hpp"
#include "gpu_matrix_ell.hpp"
#include "gpu_matrix_hyb.hpp"
#include "gpu_matrix_mcsr.hpp"
#include "gpu_matrix_bcsr.hpp"
#include "gpu_matrix_dense.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_csr.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "../../utils/allocate_free.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_csr.hpp"
#include "cuda_kernels_vector.hpp"
#include "cusparse_csr.hpp"
#include "gpu_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <hip/hip_runtime.h>
#include <hipsparse.h>

namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixCSR<ValueType>::GPUAcceleratorMatrixCSR() {

  // no default constructors
  LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixCSR<ValueType>::GPUAcceleratorMatrixCSR(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "GPUAcceleratorMatrixCSR::GPUAcceleratorMatrixCSR()",
            "constructor with local_backend");

  this->mat_.row_offset = NULL;
  this->mat_.col        = NULL;
  this->mat_.val        = NULL;
  this->set_backend(local_backend);

  this->L_mat_descr_ = 0;
  this->U_mat_descr_ = 0;

  this->L_mat_info_ = 0;
  this->U_mat_info_ = 0;

  this->mat_descr_ = 0;

  this->tmp_vec_ = NULL;

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  hipsparseStatus_t stat_t;
  
  stat_t = hipsparseCreateMatDescr(&this->mat_descr_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);
  
  stat_t = hipsparseSetMatIndexBase(this->mat_descr_, HIPSPARSE_INDEX_BASE_ZERO);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);
  
  stat_t = hipsparseSetMatType(this->mat_descr_, HIPSPARSE_MATRIX_TYPE_GENERAL);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixCSR<ValueType>::~GPUAcceleratorMatrixCSR() {

  LOG_DEBUG(this, "GPUAcceleratorMatrixCSR::~GPUAcceleratorMatrixCSR()",
            "destructor");

  this->Clear();

  hipsparseStatus_t stat_t;

  stat_t = hipsparseDestroyMatDescr(this->mat_descr_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixCSR<ValueType>");

}


template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::AllocateCSR(const int nnz, const int nrow, const int ncol) {

  assert(nnz >= 0);
  assert(ncol >= 0);
  assert(nrow >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nnz > 0) {

    allocate_gpu(nrow+1, &this->mat_.row_offset);
    allocate_gpu(nnz,    &this->mat_.col);
    allocate_gpu(nnz,    &this->mat_.val);
    
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nrow+1, mat_.row_offset);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, mat_.col);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, mat_.val);

    this->nrow_ = nrow;
    this->ncol_ = ncol;
    this->nnz_  = nnz;

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::SetDataPtrCSR(int **row_offset, int **col, ValueType **val,
                                                       const int nnz, const int nrow, const int ncol) {

  assert(*row_offset != NULL);
  assert(*col != NULL);
  assert(*val != NULL);
  assert(nnz > 0);
  assert(nrow > 0);
  assert(ncol > 0);

  this->Clear();

  this->nrow_ = nrow;
  this->ncol_ = ncol;
  this->nnz_  = nnz;

  hipDeviceSynchronize();

  this->mat_.row_offset = *row_offset;
  this->mat_.col = *col;
  this->mat_.val = *val;

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::LeaveDataPtrCSR(int **row_offset, int **col, ValueType **val) {

  assert(this->get_nrow() > 0);
  assert(this->get_ncol() > 0);
  assert(this->get_nnz() > 0);

  hipDeviceSynchronize();

  // see free_host function for details
  *row_offset = this->mat_.row_offset;
  *col = this->mat_.col;
  *val = this->mat_.val;

  this->mat_.row_offset = NULL;
  this->mat_.col = NULL;
  this->mat_.val = NULL;

  this->nrow_ = 0;
  this->ncol_ = 0;
  this->nnz_  = 0;

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    free_gpu(&this->mat_.row_offset);
    free_gpu(&this->mat_.col);
    free_gpu(&this->mat_.val);

    this->nrow_ = 0;
    this->ncol_ = 0;
    this->nnz_  = 0;

    this->LUAnalyseClear();
    this->LLAnalyseClear();

  }

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::Zeros() {

  if (this->get_nnz() > 0)
    set_to_zero_gpu(this->local_backend_.GPU_block_size,
                    this->local_backend_.GPU_max_threads,
                    this->get_nnz(), mat_.val);

  return true;

}


template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());
  
    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.row_offset,     // dst
                 cast_mat->mat_.row_offset, // src
                 (this->get_nrow()+1)*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.col,     // dst
                 cast_mat->mat_.col, // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.val,     // dst
                 cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyFromHostAsync(const HostMatrix<ValueType> &src) {

  const HostMatrixCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());
  
    if (this->get_nnz() > 0) {

      hipMemcpyAsync(this->mat_.row_offset,     // dst
                 cast_mat->mat_.row_offset, // src
                 (this->get_nrow()+1)*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(this->mat_.col,     // dst
                 cast_mat->mat_.col, // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(this->mat_.val,     // dst
                 cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixCSR<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateCSR(this->get_nnz(), this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_ncol() == dst->get_ncol());

   
    if (this->get_nnz() > 0) {

      hipMemcpy(cast_mat->mat_.row_offset, // dst
                 this->mat_.row_offset,     // src
                 (this->get_nrow()+1)*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(cast_mat->mat_.col, // dst
                 this->mat_.col,     // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(cast_mat->mat_.val, // dst
                 this->mat_.val,     // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}


template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyToHostAsync(HostMatrix<ValueType> *dst) const {

  HostMatrixCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixCSR<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateCSR(this->get_nnz(), this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_ncol() == dst->get_ncol());
   
    if (this->get_nnz() > 0) {

      hipMemcpyAsync(cast_mat->mat_.row_offset, // dst
                 this->mat_.row_offset,     // src
                 (this->get_nrow()+1)*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(cast_mat->mat_.col, // dst
                 this->mat_.col,     // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(cast_mat->mat_.val, // dst
                 this->mat_.val,     // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}


template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixCSR<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );


     assert(this->get_nnz()  == src.get_nnz());
     assert(this->get_nrow() == src.get_nrow());
     assert(this->get_ncol() == src.get_ncol());


    if (this->get_nnz() > 0) {

        hipMemcpy(this->mat_.row_offset,         // dst
                   gpu_cast_mat->mat_.row_offset, // src
                   (this->get_nrow()+1)*sizeof(int), // size
                   hipMemcpyDeviceToDevice);
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        
        hipMemcpy(this->mat_.col,         // dst
                   gpu_cast_mat->mat_.col, // src
                   this->get_nnz()*sizeof(int), // size
                   hipMemcpyDeviceToDevice);
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        
        hipMemcpy(this->mat_.val,         // dst
                   gpu_cast_mat->mat_.val, // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToDevice);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyFromAsync(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixCSR<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );


     assert(this->get_nnz()  == src.get_nnz());
     assert(this->get_nrow() == src.get_nrow());
     assert(this->get_ncol() == src.get_ncol());


    if (this->get_nnz() > 0) {

        hipMemcpy(this->mat_.row_offset,         // dst
                   gpu_cast_mat->mat_.row_offset, // src
                   (this->get_nrow()+1)*sizeof(int), // size
                   hipMemcpyDeviceToDevice);
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        
        hipMemcpy(this->mat_.col,         // dst
                   gpu_cast_mat->mat_.col, // src
                   this->get_nnz()*sizeof(int), // size
                   hipMemcpyDeviceToDevice);
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        
        hipMemcpy(this->mat_.val,         // dst
                   gpu_cast_mat->mat_.val, // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToDevice);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHostAsync(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}


template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixCSR<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixCSR<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateCSR(dst->get_nnz(), dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {

        hipMemcpy(gpu_cast_mat->mat_.row_offset, // dst
                   this->mat_.row_offset,         // src
                   (this->get_nrow()+1)*sizeof(int), // size
                   hipMemcpyDeviceToHost);
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        
        hipMemcpy(gpu_cast_mat->mat_.col, // dst
                   this->mat_.col,         // src
                   this->get_nnz()*sizeof(int), // size
                   hipMemcpyDeviceToHost);
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        
        hipMemcpy(gpu_cast_mat->mat_.val, // dst
                   this->mat_.val,         // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToHost);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyToAsync(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixCSR<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixCSR<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateCSR(dst->get_nnz(), dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());


    if (this->get_nnz() > 0) {

        hipMemcpy(gpu_cast_mat->mat_.row_offset, // dst
                   this->mat_.row_offset,         // src
                   (this->get_nrow()+1)*sizeof(int), // size
                   hipMemcpyDeviceToHost);
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        
        hipMemcpy(gpu_cast_mat->mat_.col, // dst
                   this->mat_.col,         // src
                   this->get_nnz()*sizeof(int), // size
                   hipMemcpyDeviceToHost);
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        
        hipMemcpy(gpu_cast_mat->mat_.val, // dst
                   this->mat_.val,         // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToHost);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHostAsync(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyFromCSR(const int *row_offsets, const int *col, const ValueType *val) {

  // assert CSR format
  assert(this->get_mat_format() == CSR);

  if (this->get_nnz() > 0) {

    assert(this->nrow_ > 0);
    assert(this->ncol_ > 0);

    hipMemcpy(this->mat_.row_offset,            // dst
               row_offsets,                      // src
               (this->get_nrow()+1)*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    hipMemcpy(this->mat_.col,              // dst
               col,                         // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    hipMemcpy(this->mat_.val,                    // dst
               val,                               // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::CopyToCSR(int *row_offsets, int *col, ValueType *val) const {

  // assert CSR format
  assert(this->get_mat_format() == CSR);

  if (this->get_nnz() > 0) {

    assert(this->nrow_ > 0);
    assert(this->ncol_ > 0);

    hipMemcpy(row_offsets,                      // dst
               this->mat_.row_offset,            // src
               (this->get_nrow()+1)*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    hipMemcpy(col,                         // dst
               this->mat_.col,              // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    hipMemcpy(val,                               // dst
               this->mat_.val,                    // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;

  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {
    
    this->CopyFrom(*cast_mat_csr);
    return true;
    
  }

  /*
  const GPUAcceleratorMatrixCOO<ValueType>   *cast_mat_coo;
  if ((cast_mat_coo = dynamic_cast<const GPUAcceleratorMatrixCOO<ValueType>*> (&mat)) != NULL) {

    this->Clear();


  TODO
  Allocate
  copy colmn
  copy val
  hipsparseStatus_t
      hipsparseXcoo2csr(hipsparseHandle_t handle, const int *cooRowInd,
                       int nnz, int m, int *csrRowPtr, hipsparseIndexBase_t
                       idxBase);


    FATAL_ERROR(__FILE__, __LINE__);

    this->nrow_ = cast_mat_coo->get_nrow();
    this->ncol_ = cast_mat_coo->get_ncol();
    this->nnz_  = cast_mat_coo->get_nnz();

    return true;

  }
  */

  /*
  const GPUAcceleratorMatrixDENSE<ValueType> *cast_mat_dense;
  if ((cast_mat_dense = dynamic_cast<const GPUAcceleratorMatrixDENSE<ValueType>*> (&mat)) != NULL) {

    this->Clear();
    int nnz = 0;

    FATAL_ERROR(__FILE__, __LINE__);

    this->nrow_ = cast_mat_dense->get_nrow();
    this->ncol_ = cast_mat_dense->get_ncol();
    this->nnz_  = nnz;

    return true;

  }
  */

  /*
  const GPUAcceleratorMatrixDIA<ValueType>   *cast_mat_dia;
  if ((cast_mat_dia = dynamic_cast<const GPUAcceleratorMatrixDIA<ValueType>*> (&mat)) != NULL) {

    this->Clear();
    int nnz = 0;

    FATAL_ERROR(__FILE__, __LINE__);

    this->nrow_ = cast_mat_dia->get_nrow();
    this->ncol_ = cast_mat_dia->get_ncol();
    this->nnz_  = nnz ;

    return true;

  }
  */

  /*
  const GPUAcceleratorMatrixELL<ValueType>   *cast_mat_ell;
  if ((cast_mat_ell = dynamic_cast<const GPUAcceleratorMatrixELL<ValueType>*> (&mat)) != NULL) {

    this->Clear();
    int nnz = 0;

    FATAL_ERROR(__FILE__, __LINE__);

    this->nrow_ = cast_mat_ell->get_nrow();
    this->ncol_ = cast_mat_ell->get_ncol();
    this->nnz_  = nnz ;

    return true;

  }
  */

  /*
  const GPUAcceleratorMatrixMCSR<ValueType>  *cast_mat_mcsr;
  if ((cast_mat_mcsr = dynamic_cast<const GPUAcceleratorMatrixMCSR<ValueType>*> (&mat)) != NULL) {

    this->Clear();

    FATAL_ERROR(__FILE__, __LINE__);

    this->nrow_ = cast_mat_mcsr->get_nrow();
    this->ncol_ = cast_mat_mcsr->get_ncol();
    this->nnz_  = cast_mat_mcsr->get_nnz();

    return true;

  }
  */


  /*
  const GPUAcceleratorMatrixHYB<ValueType>   *cast_mat_hyb;
  if ((cast_mat_hyb = dynamic_cast<const GPUAcceleratorMatrixHYB<ValueType>*> (&mat)) != NULL) {

    this->Clear();

    FATAL_ERROR(__FILE__, __LINE__);
    int nnz = 0;

    this->nrow_ = cast_mat_hyb->get_nrow();
    this->ncol_ = cast_mat_hyb->get_ncol();
    this->nnz_  = nnz;

    return true;

  }
  */


  return false;

}


template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::Permute( const BaseVector<int> &permutation){

  assert(&permutation != NULL);
  assert(permutation.get_size() == this->get_nrow());
  assert(permutation.get_size() == this->get_ncol());

  if (this->get_nnz() > 0) {

    int *d_nnzr     = NULL;
    int *d_nnzrPerm = NULL;
    int *d_nnzPerm  = NULL;
    int *d_offset   = NULL;
    ValueType *d_data = NULL;

    allocate_gpu<int>(this->get_nrow(), &d_nnzr);
    allocate_gpu<int>(this->get_nrow(), &d_nnzrPerm);
    allocate_gpu<int>((this->get_nrow()+1), &d_nnzPerm);
    allocate_gpu<ValueType>(this->get_nnz(), &d_data);
    allocate_gpu<int>(this->get_nnz(), &d_offset);

    const GPUAcceleratorVector<int> *cast_perm = dynamic_cast<const GPUAcceleratorVector<int>*> (&permutation);
    assert(cast_perm != NULL);

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_calc_row_nnz<int> <<< GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset, d_nnzr);
    CHECK_CUDA_ERROR(__FILE__,__LINE__);

    kernel_permute_row_nnz<int> <<< GridSize, BlockSize>>>(this->get_nrow(), d_nnzr, cast_perm->vec_, d_nnzrPerm);
    CHECK_CUDA_ERROR(__FILE__,__LINE__);

    //TODO 
    //move in extra file
    cum_sum<int, 256>(d_nnzPerm, d_nnzrPerm, this->get_nrow());

    kernel_permute_rows<ValueType, int> <<<GridSize, BlockSize>>>(
                    this->get_nrow(), 
                    this->mat_.row_offset,
                    d_nnzPerm,
                    this->mat_.col,
                    this->mat_.val,
                    cast_perm->vec_,
                    d_nnzr,
                    d_offset,
                    d_data);
    CHECK_CUDA_ERROR(__FILE__,__LINE__);

    free_gpu<int>(&this->mat_.row_offset);	

    this->mat_.row_offset = d_nnzPerm;

    int *d_buffer = NULL;
    int *h_buffer = NULL;
    int GROUP_SIZE;
    int LOCAL_SIZE;
    int FinalReduceSize;

    allocate_gpu<int>(this->local_backend_.GPU_warp * 4, &d_buffer);

    dim3 BlockSize2(this->local_backend_.GPU_block_size);
    dim3 GridSize2(this->local_backend_.GPU_warp * 4);

    GROUP_SIZE = ( size_t( ( size_t( nrow / ( this->local_backend_.GPU_warp * 4 ) ) + 1 ) 
                 / this->local_backend_.GPU_block_size ) + 1 ) * this->local_backend_.GPU_block_size;
    LOCAL_SIZE = GROUP_SIZE / this->local_backend_.GPU_block_size;

    kernel_max<int, int, 256> <<<GridSize2, BlockSize2>>> (nrow, d_nnzr, d_buffer, GROUP_SIZE, LOCAL_SIZE);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    FinalReduceSize = this->local_backend_.GPU_warp * 4;
    allocate_host(FinalReduceSize, &h_buffer);

    hipMemcpy(h_buffer, // dst
               d_buffer, // src
               FinalReduceSize*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    free_gpu<int>(&d_buffer);

    int maxnnzrow = 0;
    for (int i=0; i<FinalReduceSize; ++i)
      if (maxnnzrow < h_buffer[i])
        maxnnzrow = h_buffer[i];

    free_host(&h_buffer);

    //TODO what should happen in this case??
    if (maxnnzrow > 1024)
      return false;
    else if (maxnnzrow > 512)
      kernel_permute_cols<ValueType, int, 1024> <<<GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset,
                                                                         cast_perm->vec_, d_nnzrPerm, d_offset,
                                                                         d_data, this->mat_.col, this->mat_.val);
    else if (maxnnzrow > 256)
      kernel_permute_cols<ValueType, int, 512> <<<GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset,
                                                                         cast_perm->vec_, d_nnzrPerm, d_offset,
                                                                         d_data, this->mat_.col, this->mat_.val);
    else if (maxnnzrow > 128)
      kernel_permute_cols<ValueType, int, 256> <<<GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset,
                                                                         cast_perm->vec_, d_nnzrPerm, d_offset,
                                                                         d_data, this->mat_.col, this->mat_.val);
    else if (maxnnzrow >  64)
      kernel_permute_cols<ValueType, int, 128> <<<GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset,
                                                                         cast_perm->vec_, d_nnzrPerm, d_offset,
                                                                         d_data, this->mat_.col, this->mat_.val);
    else if (maxnnzrow >  32)
      kernel_permute_cols<ValueType, int, 64> <<<GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset,
                                                                        cast_perm->vec_, d_nnzrPerm, d_offset,
                                                                        d_data, this->mat_.col, this->mat_.val);
    else if (maxnnzrow >  16)
      kernel_permute_cols<ValueType, int, 32> <<<GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset,
                                                                        cast_perm->vec_, d_nnzrPerm, d_offset,
                                                                        d_data, this->mat_.col, this->mat_.val);
    else if (maxnnzrow >   8)
      kernel_permute_cols<ValueType, int, 16> <<<GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset,
                                                                        cast_perm->vec_, d_nnzrPerm, d_offset,
                                                                        d_data, this->mat_.col, this->mat_.val);
    else if (maxnnzrow >   4)
      kernel_permute_cols<ValueType, int, 8> <<<GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset,
                                                                       cast_perm->vec_, d_nnzrPerm, d_offset,
                                                                       d_data, this->mat_.col, this->mat_.val);
    else
      kernel_permute_cols<ValueType, int, 4> <<<GridSize, BlockSize>>>(this->get_nrow(), this->mat_.row_offset,
                                                                       cast_perm->vec_, d_nnzrPerm, d_offset,
                                                                       d_data, this->mat_.col, this->mat_.val);
    CHECK_CUDA_ERROR(__FILE__,__LINE__);

    free_gpu<int>(&d_offset);
    free_gpu<ValueType>(&d_data);
    free_gpu<int>(&d_nnzrPerm);
    free_gpu<int>(&d_nnzr);

  }

  return true;

}

template <>
void GPUAcceleratorMatrixCSR<float>::Apply(const BaseVector<float> &in, BaseVector<float> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());
    
    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in) ; 
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out) ; 
    
    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;
    const float scalar = 1.0;
    const float beta = 0.0;

    stat_t = hipsparseScsrmv(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            this->get_nrow(), this->get_ncol(), this->get_nnz(), &scalar,
                            this->mat_descr_,
                            this->mat_.val, this->mat_.row_offset, this->mat_.col,
                            cast_in->vec_, &beta,
                            cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

/*
    // Using cusparse instead...
    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);
    
    kernel_csr_spmv_scalar<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col, this->mat_.val,
                                                                      cast_in->vec_, cast_out->vec_);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
*/
  }
    
}

template <>
void GPUAcceleratorMatrixCSR<double>::Apply(const BaseVector<double> &in, BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());
    
    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in) ; 
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out) ; 
    
    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;
    const double scalar = 1.0;
    const double beta = 0.0;

    stat_t = hipsparseDcsrmv(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            this->get_nrow(), this->get_ncol(), this->get_nnz(), &scalar,
                            this->mat_descr_,
                            this->mat_.val, 
                            this->mat_.row_offset, this->mat_.col,
                            cast_in->vec_, &beta,
                            cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    /*
    // Using cusparse instead...
    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_spmv_scalar<double, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col, this->mat_.val,
                                                                      cast_in->vec_, cast_out->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);
    */
  }
    
}

template <>
void GPUAcceleratorMatrixCSR<float>::ApplyAdd(const BaseVector<float> &in, const float scalar,
                                                    BaseVector<float> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in);
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;
    const float beta = 1.0;

    stat_t = hipsparseScsrmv(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            this->get_nrow(), this->get_ncol(), this->get_nnz(), &scalar,
                            this->mat_descr_,
                            this->mat_.val, this->mat_.row_offset, this->mat_.col,
                            cast_in->vec_, &beta,
                            cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

/*
    // Using cusparse now...
    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);
    
    kernel_csr_add_spmv_scalar<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col, this->mat_.val,
                                                                          scalar, cast_in->vec_, cast_out->vec_);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);    
*/  
  }

}

template <>
void GPUAcceleratorMatrixCSR<double>::ApplyAdd(const BaseVector<double> &in, const double scalar,
                                                     BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in);
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;
    const double beta = 1.0;

    stat_t = hipsparseDcsrmv(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            this->get_nrow(), this->get_ncol(), this->get_nnz(), &scalar,
                            this->mat_descr_,
                            this->mat_.val, this->mat_.row_offset, this->mat_.col,
                            cast_in->vec_, &beta,
                            cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    /*
    // Using cusparse now...
    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);
    
    kernel_csr_add_spmv_scalar<double, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col, this->mat_.val,
                                                                          scalar, cast_in->vec_, cast_out->vec_);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);    
    */

  }

}

template <>
bool GPUAcceleratorMatrixCSR<float>::ILU0Factorize(void) {
  
  if (this->get_nnz() > 0) {

    hipsparseStatus_t stat_t;

    cusparseSolveAnalysisInfo_t infoA = 0;

    stat_t = cusparseCreateSolveAnalysisInfo(&infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseScsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseScsrilu0(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              this->get_nrow(),
                              this->mat_descr_,
                              this->mat_.val, this->mat_.row_offset, this->mat_.col,
                              infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseDestroySolveAnalysisInfo(infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<double>::ILU0Factorize(void) {

  if (this->get_nnz() > 0) {

    hipsparseStatus_t stat_t;

    cusparseSolveAnalysisInfo_t infoA = 0;

    stat_t = cusparseCreateSolveAnalysisInfo(&infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseDcsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseDcsrilu0(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              this->get_nrow(),
                              this->mat_descr_,
                              this->mat_.val, this->mat_.row_offset, this->mat_.col,
                              infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseDestroySolveAnalysisInfo(infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<float>::ICFactorize(BaseVector<float> *inv_diag) {

  if (this->get_nnz() > 0) {

    hipsparseStatus_t stat_t;

    cusparseSolveAnalysisInfo_t infoA = 0;

    stat_t = cusparseCreateSolveAnalysisInfo(&infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->mat_descr_, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseScsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle), 
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col, 
                                     infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseScsric0(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             this->get_nrow(),
                             this->mat_descr_,
                             this->mat_.val, this->mat_.row_offset, this->mat_.col,
                             infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<double>::ICFactorize(BaseVector<double> *inv_diag) {

  if (this->get_nnz() > 0) {

    hipsparseStatus_t stat_t;

    cusparseSolveAnalysisInfo_t infoA = 0;

    stat_t = cusparseCreateSolveAnalysisInfo(&infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->mat_descr_, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseDcsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle), 
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col, 
                                     infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseDcsric0(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             this->get_nrow(),
                             this->mat_descr_,
                             this->mat_.val, this->mat_.row_offset, this->mat_.col,
                             infoA);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
void GPUAcceleratorMatrixCSR<double>::LUAnalyse(void) {

    this->LUAnalyseClear();

    hipsparseStatus_t stat_t;

    // L part
    stat_t = hipsparseCreateMatDescr(&this->L_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->L_mat_descr_,HIPSPARSE_MATRIX_TYPE_GENERAL);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatIndexBase(this->L_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->L_mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->L_mat_descr_, HIPSPARSE_DIAG_TYPE_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseCreateSolveAnalysisInfo(&this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);


    // U part
    stat_t = hipsparseCreateMatDescr(&this->U_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->U_mat_descr_,HIPSPARSE_MATRIX_TYPE_GENERAL);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatIndexBase(this->U_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->U_mat_descr_, HIPSPARSE_FILL_MODE_UPPER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->U_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseCreateSolveAnalysisInfo(&this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // Analysis
    stat_t = cusparseDcsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->L_mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseDcsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                     this->get_nrow(), this->get_nnz(), 
                                     this->U_mat_descr_, 
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    assert(this->get_ncol() == this->get_nrow());
    assert(this->tmp_vec_ == NULL);
    this->tmp_vec_ = new GPUAcceleratorVector<double>(this->local_backend_);
    assert(this->tmp_vec_ != NULL);

    tmp_vec_->Allocate(this->get_nrow());

}

template <>
void GPUAcceleratorMatrixCSR<float>::LUAnalyse(void) {

    this->LUAnalyseClear();

    hipsparseStatus_t stat_t;

    // L part
    stat_t = hipsparseCreateMatDescr(&this->L_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->L_mat_descr_,HIPSPARSE_MATRIX_TYPE_GENERAL);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatIndexBase(this->L_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->L_mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->L_mat_descr_, HIPSPARSE_DIAG_TYPE_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseCreateSolveAnalysisInfo(&this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);


    // U part
    stat_t = hipsparseCreateMatDescr(&this->U_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->U_mat_descr_,HIPSPARSE_MATRIX_TYPE_GENERAL);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatIndexBase(this->U_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->U_mat_descr_, HIPSPARSE_FILL_MODE_UPPER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->U_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseCreateSolveAnalysisInfo(&this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // Analysis
    stat_t = cusparseScsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->L_mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseScsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                     this->get_nrow(), this->get_nnz(), 
                                     this->U_mat_descr_, 
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    assert(this->get_ncol() == this->get_nrow());
    assert(this->tmp_vec_ == NULL);
    this->tmp_vec_ = new GPUAcceleratorVector<float>(this->local_backend_);
    assert(this->tmp_vec_ != NULL);

    tmp_vec_->Allocate(this->get_nrow());

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::LUAnalyseClear(void) {

  hipsparseStatus_t stat_t;

  if (this->L_mat_info_ != 0) {
    stat_t = cusparseDestroySolveAnalysisInfo(this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  if (this->L_mat_descr_ != 0) {
    stat_t = hipsparseDestroyMatDescr(this->L_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  if (this->U_mat_info_ != 0) {
    stat_t = cusparseDestroySolveAnalysisInfo(this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  if (this->U_mat_descr_ != 0) {
    stat_t = hipsparseDestroyMatDescr(this->U_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  this->L_mat_descr_ = 0;
  this->U_mat_descr_ = 0;
  this->L_mat_info_ = 0;
  this->U_mat_info_ = 0;

  if (this ->tmp_vec_ != NULL) {
    delete this->tmp_vec_ ;
    this->tmp_vec_ = NULL;
  }

}

template <>
bool GPUAcceleratorMatrixCSR<float>::LUSolve(const BaseVector<float> &in, BaseVector<float> *out) const {

  if (this->get_nnz() > 0) {

    assert(this->L_mat_descr_ != 0);
    assert(this->U_mat_descr_ != 0);
    assert(this->L_mat_info_  != 0);
    assert(this->U_mat_info_  != 0);

    assert(in.  get_size()  >= 0);
    assert(out->get_size()  >= 0);
    assert(in.  get_size()  == this->get_ncol());
    assert(out->get_size()  == this->get_nrow());
    assert(this->get_ncol() == this->get_nrow());

    assert(this->tmp_vec_ != NULL);

    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in);
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;

    float one = float(1.0);

    // Solve L
    stat_t = cusparseScsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->L_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->L_mat_info_,
                                  cast_in->vec_,
                                  tmp_vec_->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // Solve U
    stat_t = cusparseScsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->U_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->U_mat_info_,
                                  tmp_vec_->vec_,
                                  cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<double>::LUSolve(const BaseVector<double> &in, BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(this->L_mat_descr_ != 0);
    assert(this->U_mat_descr_ != 0);
    assert(this->L_mat_info_  != 0);
    assert(this->U_mat_info_  != 0);

    assert(in.  get_size()  >= 0);
    assert(out->get_size()  >= 0);
    assert(in.  get_size()  == this->get_ncol());
    assert(out->get_size()  == this->get_nrow());
    assert(this->get_ncol() == this->get_nrow());

    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in);
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;

    double one = double(1.0);

    // Solve L
    stat_t = cusparseDcsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->L_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->L_mat_info_,
                                  cast_in->vec_,
                                  this->tmp_vec_->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // Solve U
    stat_t = cusparseDcsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->U_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->U_mat_info_,
                                  this->tmp_vec_->vec_,
                                  cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
void GPUAcceleratorMatrixCSR<double>::LLAnalyse(void) {

    this->LLAnalyseClear();

    hipsparseStatus_t stat_t;

    // L part
    stat_t = hipsparseCreateMatDescr(&this->L_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->L_mat_descr_,HIPSPARSE_MATRIX_TYPE_TRIANGULAR);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatIndexBase(this->L_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->L_mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->L_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseCreateSolveAnalysisInfo(&this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // U part
    stat_t = hipsparseCreateMatDescr(&this->U_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->U_mat_descr_,HIPSPARSE_MATRIX_TYPE_TRIANGULAR);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatIndexBase(this->U_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->U_mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->U_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseCreateSolveAnalysisInfo(&this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // Analysis
    stat_t = cusparseDcsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->L_mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseDcsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->U_mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    assert(this->get_ncol() == this->get_nrow());
    assert(this->tmp_vec_ == NULL);
    this->tmp_vec_ = new GPUAcceleratorVector<double>(this->local_backend_);
    assert(this->tmp_vec_ != NULL);

    tmp_vec_->Allocate(this->get_nrow());

}

template <>
void GPUAcceleratorMatrixCSR<float>::LLAnalyse(void) {

    this->LLAnalyseClear();

    hipsparseStatus_t stat_t;

    // L part
    stat_t = hipsparseCreateMatDescr(&this->L_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->L_mat_descr_,HIPSPARSE_MATRIX_TYPE_TRIANGULAR);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatIndexBase(this->L_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->L_mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->L_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseCreateSolveAnalysisInfo(&this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // U part
    stat_t = hipsparseCreateMatDescr(&this->U_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatType(this->U_mat_descr_,HIPSPARSE_MATRIX_TYPE_TRIANGULAR);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatIndexBase(this->U_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatFillMode(this->U_mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = hipsparseSetMatDiagType(this->U_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseCreateSolveAnalysisInfo(&this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // Analysis
    stat_t = cusparseScsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->L_mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    stat_t = cusparseScsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                     HIPSPARSE_OPERATION_TRANSPOSE,
                                     this->get_nrow(), this->get_nnz(),
                                     this->U_mat_descr_,
                                     this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                     this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    assert(this->get_ncol() == this->get_nrow());
    assert(this->tmp_vec_ == NULL);
    this->tmp_vec_ = new GPUAcceleratorVector<float>(this->local_backend_);
    assert(this->tmp_vec_ != NULL);

    tmp_vec_->Allocate(this->get_nrow());

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::LLAnalyseClear(void) {

  hipsparseStatus_t stat_t;

  if (this->L_mat_info_ != 0) {
    stat_t = cusparseDestroySolveAnalysisInfo(this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  if (this->L_mat_descr_ != 0) {
    stat_t = hipsparseDestroyMatDescr(this->L_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  if (this->U_mat_info_ != 0) {
    stat_t = cusparseDestroySolveAnalysisInfo(this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  if (this->U_mat_descr_ != 0) {
    stat_t = hipsparseDestroyMatDescr(this->U_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  this->L_mat_descr_ = 0;
  this->U_mat_descr_ = 0;
  this->L_mat_info_ = 0;
  this->U_mat_info_ = 0;

  if (this ->tmp_vec_ != NULL) {
    delete this->tmp_vec_ ;
    this->tmp_vec_ = NULL;
  }
    

}

template <>
bool GPUAcceleratorMatrixCSR<double>::LLSolve(const BaseVector<double> &in, BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(this->L_mat_descr_ != 0);
    assert(this->U_mat_descr_ != 0);
    assert(this->L_mat_info_  != 0);
    assert(this->U_mat_info_  != 0);

    assert(in.  get_size()  >= 0);
    assert(out->get_size()  >= 0);
    assert(in.  get_size()  == this->get_ncol());
    assert(out->get_size()  == this->get_nrow());
    assert(this->get_ncol() == this->get_nrow());

    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in);
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;

    double one = double(1.0);

    // Solve L
    stat_t = cusparseDcsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->L_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->L_mat_info_,
                                  cast_in->vec_,
                                  this->tmp_vec_->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // Solve U
    stat_t = cusparseDcsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->U_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->U_mat_info_,
                                  this->tmp_vec_->vec_,
                                  cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<float>::LLSolve(const BaseVector<float> &in, BaseVector<float> *out) const {

  if (this->get_nnz() > 0) {

    assert(this->L_mat_descr_ != 0);
    assert(this->U_mat_descr_ != 0);
    assert(this->L_mat_info_  != 0);
    assert(this->U_mat_info_  != 0);

    assert(in.  get_size()  >= 0);
    assert(out->get_size()  >= 0);
    assert(in.  get_size()  == this->get_ncol());
    assert(out->get_size()  == this->get_nrow());
    assert(this->get_ncol() == this->get_nrow());

    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in);
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;

    float one = float(1.0);

    // Solve L
    stat_t = cusparseScsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->L_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->L_mat_info_,
                                  cast_in->vec_,
                                  this->tmp_vec_->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

    // Solve U
    stat_t = cusparseScsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->U_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->U_mat_info_,
                                  this->tmp_vec_->vec_,
                                  cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::LLSolve(const BaseVector<ValueType> &in, const BaseVector<ValueType> &inv_diag,
                                                 BaseVector<ValueType> *out) const {

  return LLSolve(in, out);

}

template <>
void GPUAcceleratorMatrixCSR<double>::LAnalyse(const bool diag_unit) {

  hipsparseStatus_t stat_t;

  // L part
  stat_t = hipsparseCreateMatDescr(&this->L_mat_descr_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatType(this->L_mat_descr_,HIPSPARSE_MATRIX_TYPE_GENERAL);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatIndexBase(this->L_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatFillMode(this->L_mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  if (diag_unit == true) {

    stat_t = hipsparseSetMatDiagType(this->L_mat_descr_, HIPSPARSE_DIAG_TYPE_UNIT);

  } else {

    stat_t = hipsparseSetMatDiagType(this->L_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);

  }

  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = cusparseCreateSolveAnalysisInfo(&this->L_mat_info_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  // Analysis
  stat_t = cusparseDcsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                   this->get_nrow(), this->get_nnz(),
                                   this->L_mat_descr_,
                                   this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                   this->L_mat_info_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

}

template <>
void GPUAcceleratorMatrixCSR<float>::LAnalyse(const bool diag_unit) {

  hipsparseStatus_t stat_t;

  // L part
  stat_t = hipsparseCreateMatDescr(&this->L_mat_descr_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatType(this->L_mat_descr_,HIPSPARSE_MATRIX_TYPE_GENERAL);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatIndexBase(this->L_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatFillMode(this->L_mat_descr_, HIPSPARSE_FILL_MODE_LOWER);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  if (diag_unit == true) {

    stat_t = hipsparseSetMatDiagType(this->L_mat_descr_, HIPSPARSE_DIAG_TYPE_UNIT);

  } else {

    stat_t = hipsparseSetMatDiagType(this->L_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);

  }

  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 

  stat_t = cusparseCreateSolveAnalysisInfo(&this->L_mat_info_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  // Analysis
  stat_t = cusparseScsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                   this->get_nrow(), this->get_nnz(),
                                   this->L_mat_descr_,
                                   this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                   this->L_mat_info_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

}

template <>
void GPUAcceleratorMatrixCSR<double>::UAnalyse(const bool diag_unit) {

  hipsparseStatus_t stat_t;

  // U upart
  stat_t = hipsparseCreateMatDescr(&this->U_mat_descr_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatType(this->U_mat_descr_,HIPSPARSE_MATRIX_TYPE_GENERAL);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatIndexBase(this->U_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatFillMode(this->U_mat_descr_, HIPSPARSE_FILL_MODE_UPPER);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  if (diag_unit == true) {

    stat_t = hipsparseSetMatDiagType(this->U_mat_descr_, HIPSPARSE_DIAG_TYPE_UNIT);

  } else {

    stat_t = hipsparseSetMatDiagType(this->U_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);

  }

  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = cusparseCreateSolveAnalysisInfo(&this->U_mat_info_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  // Analysis
  stat_t = cusparseDcsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                   this->get_nrow(), this->get_nnz(),
                                   this->U_mat_descr_,
                                   this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                   this->U_mat_info_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

}

template <>
void GPUAcceleratorMatrixCSR<float>::UAnalyse(const bool diag_unit) {

  hipsparseStatus_t stat_t;

  // U part
  stat_t = hipsparseCreateMatDescr(&this->U_mat_descr_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatType(this->U_mat_descr_,HIPSPARSE_MATRIX_TYPE_GENERAL);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatIndexBase(this->U_mat_descr_,HIPSPARSE_INDEX_BASE_ZERO);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseSetMatFillMode(this->U_mat_descr_, HIPSPARSE_FILL_MODE_UPPER);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  if (diag_unit == true) {

    stat_t = hipsparseSetMatDiagType(this->U_mat_descr_, HIPSPARSE_DIAG_TYPE_UNIT);

  } else {

    stat_t = hipsparseSetMatDiagType(this->U_mat_descr_, HIPSPARSE_DIAG_TYPE_NON_UNIT);

  }

  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = cusparseCreateSolveAnalysisInfo(&this->U_mat_info_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  // Analysis
  stat_t = cusparseScsrsv_analysis(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                   this->get_nrow(), this->get_nnz(),
                                   this->U_mat_descr_,
                                   this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                   this->U_mat_info_);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::LAnalyseClear(void) {

  hipsparseStatus_t stat_t;

  if (this->L_mat_info_ != 0) {
    stat_t = cusparseDestroySolveAnalysisInfo(this->L_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  if (this->L_mat_descr_ != 0) {
    stat_t = hipsparseDestroyMatDescr(this->L_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  this->L_mat_descr_ = 0;
  this->L_mat_info_ = 0;

}

template <typename ValueType>
void GPUAcceleratorMatrixCSR<ValueType>::UAnalyseClear(void) {

  hipsparseStatus_t stat_t;

  if (this->U_mat_info_ != 0) {
    stat_t = cusparseDestroySolveAnalysisInfo(this->U_mat_info_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  if (this->U_mat_descr_ != 0) {
    stat_t = hipsparseDestroyMatDescr(this->U_mat_descr_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__); 
  }

  this->U_mat_descr_ = 0;
  this->U_mat_info_ = 0;

}

template <>
bool GPUAcceleratorMatrixCSR<double>::LSolve(const BaseVector<double> &in, BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(this->L_mat_descr_ != 0);
    assert(this->U_mat_descr_ != 0);
    assert(this->L_mat_info_  != 0);
    assert(this->U_mat_info_  != 0);

    assert(in.  get_size()  >= 0);
    assert(out->get_size()  >= 0);
    assert(in.  get_size()  == this->get_ncol());
    assert(out->get_size()  == this->get_nrow());
    assert(this->get_ncol() == this->get_nrow());

    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in);
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;

    double one = double(1.0);

    // Solve L
    stat_t = cusparseDcsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->L_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->L_mat_info_,
                                  cast_in->vec_,
                                  cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<float>::LSolve(const BaseVector<float> &in, BaseVector<float> *out) const {

  if (this->get_nnz() > 0) {

    assert(this->L_mat_descr_ != 0);
    assert(this->U_mat_descr_ != 0);
    assert(this->L_mat_info_  != 0);
    assert(this->U_mat_info_  != 0);

    assert(in.  get_size()  >= 0);
    assert(out->get_size()  >= 0);
    assert(in.  get_size()  == this->get_ncol());
    assert(out->get_size()  == this->get_nrow());
    assert(this->get_ncol() == this->get_nrow());

    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in);
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;

    float one = float(1.0);

    // Solve L
    stat_t = cusparseScsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->L_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->L_mat_info_,
                                  cast_in->vec_,
                                  cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<double>::USolve(const BaseVector<double> &in, BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(this->L_mat_descr_ != 0);
    assert(this->U_mat_descr_ != 0);
    assert(this->L_mat_info_  != 0);
    assert(this->U_mat_info_  != 0);

    assert(in.  get_size()  >= 0);
    assert(out->get_size()  >= 0);
    assert(in.  get_size()  == this->get_ncol());
    assert(out->get_size()  == this->get_nrow());
    assert(this->get_ncol() == this->get_nrow()); 

    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in);
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;

    double one = double(1.0);

    // Solve U
    stat_t = cusparseDcsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->U_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->U_mat_info_,
                                  cast_in->vec_,
                                  cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<float>::USolve(const BaseVector<float> &in, BaseVector<float> *out) const {

  if (this->get_nnz() > 0) {

    assert(this->L_mat_descr_ != 0);
    assert(this->U_mat_descr_ != 0);
    assert(this->L_mat_info_  != 0);
    assert(this->U_mat_info_  != 0);

    assert(in.  get_size()  >= 0);
    assert(out->get_size()  >= 0);
    assert(in.  get_size()  == this->get_ncol());
    assert(out->get_size()  == this->get_nrow());
    assert(this->get_ncol() == this->get_nrow());

    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in);
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipsparseStatus_t stat_t;

    float one = float(1.0);

    // Solve U
    stat_t = cusparseScsrsv_solve(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  this->get_nrow(),
                                  &one,
                                  this->U_mat_descr_,
                                  this->mat_.val, this->mat_.row_offset, this->mat_.col,
                                  this->U_mat_info_,
                                  cast_in->vec_,
                                  cast_out->vec_);
    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ExtractDiagonal(BaseVector<ValueType> *vec_diag) const {

  if (this->get_nnz() > 0)  {

    assert(vec_diag != NULL);
    assert(vec_diag->get_size() == this->get_nrow());

    GPUAcceleratorVector<ValueType> *cast_vec_diag  = dynamic_cast<GPUAcceleratorVector<ValueType>*> (vec_diag);

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_extract_diag<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col, this->mat_.val,
                                                                       cast_vec_diag->vec_);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ExtractInverseDiagonal(BaseVector<ValueType> *vec_inv_diag) const {

  if (this->get_nnz() > 0) {

    assert(vec_inv_diag != NULL);
    assert(vec_inv_diag->get_size() == this->get_nrow());

    GPUAcceleratorVector<ValueType> *cast_vec_inv_diag  = dynamic_cast<GPUAcceleratorVector<ValueType>*> (vec_inv_diag);

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_extract_inv_diag<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col,
                                                                           this->mat_.val, cast_vec_inv_diag->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ExtractSubMatrix(const int row_offset,
                                                          const int col_offset,
                                                          const int row_size,
                                                          const int col_size,
                                                          BaseMatrix<ValueType> *mat) const {
  assert(mat != NULL);

  assert(row_offset >= 0);
  assert(col_offset >= 0);

  assert(this->get_nrow() > 0);
  assert(this->get_ncol() > 0);

  GPUAcceleratorMatrixCSR<ValueType> *cast_mat  = dynamic_cast<GPUAcceleratorMatrixCSR<ValueType>*> (mat) ; 
  assert(cast_mat != NULL);

  int mat_nnz = 0;

  int *row_nnz = NULL;  
  //int *red_row_nnz  (int *) malloc(sizeof(int)*(row_size+1));
  int *sub_nnz = NULL;
  allocate_gpu<int>(row_size+1, &sub_nnz);
  allocate_gpu(row_size+1, &row_nnz);

  // compute the nnz per row in the new matrix

  dim3 BlockSize(this->local_backend_.GPU_block_size);
  dim3 GridSize(row_size / this->local_backend_.GPU_block_size + 1);
  
  kernel_csr_extract_submatrix_row_nnz<ValueType, int> <<<GridSize, BlockSize>>> (this->mat_.row_offset, this->mat_.col, this->mat_.val,
                                                                                  row_offset, col_offset, 
                                                                                  row_size, col_size, 
                                                                                  row_nnz);
    
  CHECK_CUDA_ERROR(__FILE__, __LINE__);      

  // compute the new nnz by reduction 
  
  
  // CPU reduction
  /*
  hipMemcpy(red_row_nnz, // dst
             row_nnz,  // src
             (row_size+1)*sizeof(int), // size
             hipMemcpyDeviceToHost);

  int sum=0;
  for (int i=0; i<row_size; ++i) {
    int tmp = red_row_nnz[i];
    red_row_nnz[i] = sum;
    sum += tmp;
  }

  mat_nnz = red_row_nnz[row_size] = sum ;
  */

  //TODO
  //move in extra file
  cum_sum<int, 256>(sub_nnz, row_nnz, row_size);
  
  hipMemcpy(&mat_nnz, &sub_nnz[row_size],
             sizeof(int), hipMemcpyDeviceToHost);

  // not empty submatrix
  if (mat_nnz > 0) {

    cast_mat->AllocateCSR(mat_nnz, row_size, col_size);

    // part of the CPU reduction section
    /*
    hipMemcpy(cast_mat->mat_.row_offset, // dst
               red_row_nnz,  // src
               (row_size+1)*sizeof(int), // size
               hipMemcpyHostToDevice);
    */
    
    free_gpu<int>(&cast_mat->mat_.row_offset);
    cast_mat->mat_.row_offset = sub_nnz;
    // copying the sub matrix
    
    kernel_csr_extract_submatrix_copy<ValueType, int> <<<GridSize, BlockSize>>> (this->mat_.row_offset, this->mat_.col, this->mat_.val,
                                                                                 row_offset, col_offset, 
                                                                                 row_size, col_size,
                                                                                 cast_mat->mat_.row_offset, cast_mat->mat_.col, cast_mat->mat_.val);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      

  }

  free_gpu(&row_nnz);

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ExtractL(BaseMatrix<ValueType> *L) const {
  
  assert(L != NULL);
  
  assert(this->get_nrow() > 0);
  assert(this->get_ncol() > 0);
  
  GPUAcceleratorMatrixCSR<ValueType> *cast_L = dynamic_cast<GPUAcceleratorMatrixCSR<ValueType>*> (L);
  
  assert(cast_L != NULL);
  
  cast_L->Clear();
  
  // compute nnz per row
  int nrow = this->get_nrow();
  
  allocate_gpu<int>(nrow+1, &cast_L->mat_.row_offset);
  
  dim3 BlockSize(this->local_backend_.GPU_block_size);
  dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);
  
  
  kernel_csr_slower_nnz_per_row<int> <<<GridSize, BlockSize>>>(nrow, this->mat_.row_offset,
                                                              this->mat_.col, cast_L->mat_.row_offset+1);
  CHECK_CUDA_ERROR(__FILE__,__LINE__);
  
  // partial sum row_nnz to obtain row_offset vector
  // TODO currently performing partial sum on host
  int *h_buffer = NULL;
  allocate_host(nrow+1, &h_buffer);
  hipMemcpy(h_buffer+1, // dst
             cast_L->mat_.row_offset+1, // src
             nrow*sizeof(int), // size
             hipMemcpyDeviceToHost);
  
  h_buffer[0] = 0;
  for (int i=1; i<nrow+1; ++i)
    h_buffer[i] += h_buffer[i-1];
  
  int nnz_L = h_buffer[nrow];
  
  hipMemcpy(cast_L->mat_.row_offset, // dst
             h_buffer, // src
             (nrow+1)*sizeof(int), // size
             hipMemcpyHostToDevice);
  
  free_host(&h_buffer);
  // end TODO
  
  // allocate lower triangular part structure
  allocate_gpu<int>(nnz_L, &cast_L->mat_.col);
  allocate_gpu<ValueType>(nnz_L, &cast_L->mat_.val);
  
  // fill lower triangular part
  kernel_csr_extract_l_triangular<ValueType, int> <<<GridSize, BlockSize>>>(nrow, this->mat_.row_offset,
                                                                            this->mat_.col, this->mat_.val,
                                                                            cast_L->mat_.row_offset,
                                                                            cast_L->mat_.col,
                                                                            cast_L->mat_.val);
  CHECK_CUDA_ERROR(__FILE__,__LINE__);
  
  cast_L->nrow_ = this->get_nrow();
  cast_L->ncol_ = this->get_ncol();
  cast_L->nnz_ = nnz_L;
  
  return true;
  
}


template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ExtractLDiagonal(BaseMatrix<ValueType> *L) const {

  assert(L != NULL);
  
  assert(this->get_nrow() > 0);
  assert(this->get_ncol() > 0);
  
  GPUAcceleratorMatrixCSR<ValueType> *cast_L = dynamic_cast<GPUAcceleratorMatrixCSR<ValueType>*> (L);
  
  assert(cast_L != NULL);
  
  cast_L->Clear();
  
  // compute nnz per row
  int nrow = this->get_nrow();
  
  allocate_gpu<int>(nrow+1, &cast_L->mat_.row_offset);
  
  dim3 BlockSize(this->local_backend_.GPU_block_size);
  dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

  kernel_csr_lower_nnz_per_row<int> <<<GridSize, BlockSize>>>(nrow, this->mat_.row_offset,
                                                              this->mat_.col, cast_L->mat_.row_offset+1);
  CHECK_CUDA_ERROR(__FILE__,__LINE__);

  // partial sum row_nnz to obtain row_offset vector
  // TODO currently performing partial sum on host
  int *h_buffer = NULL;
  allocate_host(nrow+1, &h_buffer);
  hipMemcpy(h_buffer+1, // dst
             cast_L->mat_.row_offset+1, // src
             nrow*sizeof(int), // size
             hipMemcpyDeviceToHost);

  h_buffer[0] = 0;
  for (int i=1; i<nrow+1; ++i)
    h_buffer[i] += h_buffer[i-1];
  
  int nnz_L = h_buffer[nrow];

  hipMemcpy(cast_L->mat_.row_offset, // dst
             h_buffer, // src
             (nrow+1)*sizeof(int), // size
             hipMemcpyHostToDevice);
  
  free_host(&h_buffer);
  // end TODO
  
  // allocate lower triangular part structure
  allocate_gpu<int>(nnz_L, &cast_L->mat_.col);
  allocate_gpu<ValueType>(nnz_L, &cast_L->mat_.val);
  
  // fill lower triangular part
  kernel_csr_extract_l_triangular<ValueType, int> <<<GridSize, BlockSize>>>(nrow, this->mat_.row_offset,
                                                                            this->mat_.col, this->mat_.val,
                                                                            cast_L->mat_.row_offset,
                                                                            cast_L->mat_.col,
                                                                            cast_L->mat_.val);
  CHECK_CUDA_ERROR(__FILE__,__LINE__);
  
  cast_L->nrow_ = this->get_nrow();
  cast_L->ncol_ = this->get_ncol();
  cast_L->nnz_ = nnz_L;
  
  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ExtractU(BaseMatrix<ValueType> *U) const {
  
  assert(U != NULL);
  
  assert(this->get_nrow() > 0);
  assert(this->get_ncol() > 0);
  
  GPUAcceleratorMatrixCSR<ValueType> *cast_U = dynamic_cast<GPUAcceleratorMatrixCSR<ValueType>*> (U);
  
  assert(cast_U != NULL);
  
  cast_U->Clear();
  
  // compute nnz per row
  int nrow = this->get_nrow();
  
  allocate_gpu<int>(nrow+1, &cast_U->mat_.row_offset);
  
  dim3 BlockSize(this->local_backend_.GPU_block_size);
  dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);
  
  
  kernel_csr_supper_nnz_per_row<int> <<<GridSize, BlockSize>>>(nrow, this->mat_.row_offset,
                                                              this->mat_.col, cast_U->mat_.row_offset+1);
  CHECK_CUDA_ERROR(__FILE__,__LINE__);
  
  // partial sum row_nnz to obtain row_offset vector
  // TODO currently performing partial sum on host
  int *h_buffer = NULL;
  allocate_host(nrow+1, &h_buffer);
  hipMemcpy(h_buffer+1, // dst
             cast_U->mat_.row_offset+1, // src
             nrow*sizeof(int), // size
             hipMemcpyDeviceToHost);
  
  h_buffer[0] = 0;
  for (int i=1; i<nrow+1; ++i)
    h_buffer[i] += h_buffer[i-1];
  
  int nnz_L = h_buffer[nrow];
  
  hipMemcpy(cast_U->mat_.row_offset, // dst
             h_buffer, // src
             (nrow+1)*sizeof(int), // size
             hipMemcpyHostToDevice);
  
  free_host(&h_buffer);
  // end TODO
  
  // allocate lower triangular part structure
  allocate_gpu<int>(nnz_L, &cast_U->mat_.col);
  allocate_gpu<ValueType>(nnz_L, &cast_U->mat_.val);
  
  // fill upper triangular part
  kernel_csr_extract_u_triangular<ValueType, int> <<<GridSize, BlockSize>>>(nrow, this->mat_.row_offset,
                                                                            this->mat_.col, this->mat_.val,
                                                                            cast_U->mat_.row_offset,
                                                                            cast_U->mat_.col,
                                                                            cast_U->mat_.val);
  CHECK_CUDA_ERROR(__FILE__,__LINE__);
  
  cast_U->nrow_ = this->get_nrow();
  cast_U->ncol_ = this->get_ncol();
  cast_U->nnz_ = nnz_L;
  
  return true;
  
}


template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ExtractUDiagonal(BaseMatrix<ValueType> *U) const {

  assert(U != NULL);
  
  assert(this->get_nrow() > 0);
  assert(this->get_ncol() > 0);
  
  GPUAcceleratorMatrixCSR<ValueType> *cast_U = dynamic_cast<GPUAcceleratorMatrixCSR<ValueType>*> (U);
  
  assert(cast_U != NULL);
  
  cast_U->Clear();
  
  // compute nnz per row
  int nrow = this->get_nrow();
  
  allocate_gpu<int>(nrow+1, &cast_U->mat_.row_offset);
  
  dim3 BlockSize(this->local_backend_.GPU_block_size);
  dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);
  
  
  kernel_csr_upper_nnz_per_row<int> <<<GridSize, BlockSize>>>(nrow, this->mat_.row_offset,
                                                              this->mat_.col, cast_U->mat_.row_offset+1);
  CHECK_CUDA_ERROR(__FILE__,__LINE__);
  
  // partial sum row_nnz to obtain row_offset vector
  // TODO currently performing partial sum on host
  int *h_buffer = NULL;
  allocate_host(nrow+1, &h_buffer);
  hipMemcpy(h_buffer+1, // dst
             cast_U->mat_.row_offset+1, // src
             nrow*sizeof(int), // size
             hipMemcpyDeviceToHost);

  h_buffer[0] = 0;
  for (int i=1; i<nrow+1; ++i)
    h_buffer[i] += h_buffer[i-1];
  
  int nnz_L = h_buffer[nrow];

  hipMemcpy(cast_U->mat_.row_offset, // dst
             h_buffer, // src
             (nrow+1)*sizeof(int), // size
             hipMemcpyHostToDevice);
  
  free_host(&h_buffer);
  // end TODO
  
  // allocate lower triangular part structure
  allocate_gpu<int>(nnz_L, &cast_U->mat_.col);
  allocate_gpu<ValueType>(nnz_L, &cast_U->mat_.val);
  
  // fill lower triangular part
  kernel_csr_extract_u_triangular<ValueType, int> <<<GridSize, BlockSize>>>(nrow, this->mat_.row_offset,
                                                                            this->mat_.col, this->mat_.val,
                                                                            cast_U->mat_.row_offset,
                                                                            cast_U->mat_.col,
                                                                            cast_U->mat_.val);
  CHECK_CUDA_ERROR(__FILE__,__LINE__);
  
  cast_U->nrow_ = this->get_nrow();
  cast_U->ncol_ = this->get_ncol();
  cast_U->nnz_ = nnz_L;
  
  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::MaximalIndependentSet(int &size,
                                                               BaseVector<int> *permutation) const {
  assert(permutation != NULL);
  GPUAcceleratorVector<int> *cast_perm = dynamic_cast<GPUAcceleratorVector<int>*> (permutation);
  assert(cast_perm != NULL);
  assert(this->get_nrow() == this->get_ncol());

  int *h_row_offset = NULL;
  int *h_col = NULL;

  allocate_host(this->get_nrow()+1, &h_row_offset);
  allocate_host(this->get_nnz(), &h_col);

  hipMemcpy(h_row_offset, this->mat_.row_offset, (this->get_nrow()+1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_col, this->mat_.col, this->get_nnz()*sizeof(int), hipMemcpyDeviceToHost);

  int *mis = NULL;
  allocate_host(this->get_nrow(), &mis);
  memset(mis, 0, sizeof(int)*this->get_nrow());

  size = 0 ;

  for (int ai=0; ai<this->get_nrow(); ++ai) {

    if (mis[ai] == 0) {

      // set the node
      mis[ai] = 1;
      ++size ;

      //remove all nbh nodes (without diagonal)
      for (int aj=h_row_offset[ai]; aj<h_row_offset[ai+1]; ++aj)
        if (ai != h_col[aj])
          mis[h_col[aj]] = -1 ;
      
    }
  }

  int *h_perm = NULL;
  allocate_host(this->get_nrow(), &h_perm);

  int pos = 0;
  for (int ai=0; ai<this->get_nrow(); ++ai) {

    if (mis[ai] == 1) {

      h_perm[ai] = pos;
      ++pos;

    } else {

      h_perm[ai] = size + ai - pos;

    }

  }
  
  // Check the permutation
  //
  //  for (int ai=0; ai<this->get_nrow(); ++ai) {
  //    assert( h_perm[ai] >= 0 );
  //    assert( h_perm[ai] < this->get_nrow() );
  //  }


  cast_perm->Allocate(this->get_nrow());
  hipMemcpy(cast_perm->vec_, h_perm, permutation->get_size()*sizeof(int), hipMemcpyHostToDevice);

  free_host(&h_row_offset);
  free_host(&h_col);

  free_host(&h_perm);
  free_host(&mis);

  return true;
}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::MultiColoring(int &num_colors,
                                                       int **size_colors,
                                                       BaseVector<int> *permutation) const {

  assert(permutation != NULL);
  GPUAcceleratorVector<int> *cast_perm = dynamic_cast<GPUAcceleratorVector<int>*> (permutation);
  assert(cast_perm != NULL);

  // node colors (init value = 0 i.e. no color)
  int *color = NULL;
  int *h_row_offset = NULL;
  int *h_col = NULL;
  int size = this->get_nrow();
  allocate_host(size, &color);
  allocate_host(this->get_nrow()+1, &h_row_offset);
  allocate_host(this->get_nnz(), &h_col);

  hipMemcpy(h_row_offset, this->mat_.row_offset, (this->get_nrow()+1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_col, this->mat_.col, this->get_nnz()*sizeof(int), hipMemcpyDeviceToHost);

  memset(color, 0, size*sizeof(int));
  num_colors = 0;
  std::vector<bool> row_col;

  for (int ai=0; ai<this->get_nrow(); ++ai) {
    color[ai] = 1;
    row_col.clear();
    row_col.assign(num_colors+2, false);

    for (int aj=h_row_offset[ai]; aj<h_row_offset[ai+1]; ++aj)
      if (ai != h_col[aj])
        row_col[color[h_col[aj]]] = true;

    for (int aj=h_row_offset[ai]; aj<h_row_offset[ai+1]; ++aj)
      if (row_col[color[ai]] == true)
        ++color[ai];

    if (color[ai] > num_colors)
      num_colors = color[ai];

  }

  free_host(&h_row_offset);
  free_host(&h_col);

  allocate_host(num_colors, size_colors);
  set_to_zero_host(num_colors, *size_colors);

  int *offsets_color = NULL;
  allocate_host(num_colors, &offsets_color);
  memset(offsets_color, 0, sizeof(int)*num_colors);

  for (int i=0; i<this->get_nrow(); ++i) 
    ++(*size_colors)[color[i]-1];

  int total=0;
  for (int i=1; i<num_colors; ++i) {

    total += (*size_colors)[i-1];
    offsets_color[i] = total; 
    //   LOG_INFO("offsets = " << total);

  }

  int *h_perm = NULL;
  allocate_host(this->get_nrow(), &h_perm);

  for (int i=0; i<this->get_nrow(); ++i) {

    h_perm[i] = offsets_color[ color[i]-1 ] ;
    ++offsets_color[color[i]-1];

  }

  cast_perm->Allocate(this->get_nrow());
  hipMemcpy(cast_perm->vec_, h_perm, permutation->get_size()*sizeof(int), hipMemcpyHostToDevice);

  free_host(&h_perm);
  free_host(&color);
  free_host(&offsets_color);

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<double>::Scale(const double alpha) {

  if (this->get_nnz() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDscal(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                         this->get_nnz(), &alpha,
                         this->mat_.val, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<float>::Scale(const float alpha) {

  if (this->get_nnz() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasSscal(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                         this->get_nnz(), &alpha,
                         this->mat_.val, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ScaleDiagonal(const ValueType alpha) {

  if (this->get_nnz() > 0) {

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_scale_diagonal<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col,
                                                                         alpha, this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ScaleOffDiagonal(const ValueType alpha) {

  if (this->get_nnz() > 0) {

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_scale_offdiagonal<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col,
                                                                            alpha, this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::AddScalarDiagonal(const ValueType alpha) {

  if (this->get_nnz() > 0) {

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_add_diagonal<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col,
                                                                       alpha, this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::AddScalarOffDiagonal(const ValueType alpha) {

  if (this->get_nnz() > 0) {

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_add_offdiagonal<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col,
                                                                       alpha, this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::AddScalar(const ValueType alpha) {

  if (this->get_nnz() > 0) {

    int nnz = this->get_nnz();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nnz / this->local_backend_.GPU_block_size + 1);

    kernel_buffer_addscalar<ValueType, int> <<<GridSize, BlockSize>>> (nnz, alpha, this->mat_.val);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::DiagonalMatrixMultR(const BaseVector<ValueType> &diag) {

  assert(diag.get_size() == this->get_ncol());

  const GPUAcceleratorVector<ValueType> *cast_diag = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&diag);
  assert(cast_diag!= NULL);

  if (this->get_nnz() > 0) {

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_diagmatmult_r<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset, this->mat_.col,
                                                                      cast_diag->vec_, this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::DiagonalMatrixMultL(const BaseVector<ValueType> &diag) {

  assert(diag.get_size() == this->get_ncol());

  const GPUAcceleratorVector<ValueType> *cast_diag = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&diag);
  assert(cast_diag!= NULL);

  if (this->get_nnz() > 0) {

    int nrow = this->get_nrow();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_diagmatmult_l<ValueType, int> <<<GridSize, BlockSize>>> (nrow, this->mat_.row_offset,
                                                                      cast_diag->vec_, this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::MatMatMult(const BaseMatrix<ValueType> &A, const BaseMatrix<ValueType> &B) {

  assert(A.get_ncol() == B.get_nrow());
  assert(A.get_nrow() > 0);
  assert(B.get_ncol() > 0);
  assert(B.get_nrow() > 0);

  const GPUAcceleratorMatrixCSR<ValueType> *cast_mat_A = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&A);
  const GPUAcceleratorMatrixCSR<ValueType> *cast_mat_B = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&B);
  assert(cast_mat_A != NULL);
  assert(cast_mat_B != NULL);

  this->Clear();

  int m = cast_mat_A->get_nrow();
  int n = cast_mat_B->get_ncol();
  int k = cast_mat_B->get_nrow();
  int nnzC = 0;

  allocate_gpu(m+1, &this->mat_.row_offset);
  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  hipsparseStatus_t stat_t;

  stat_t = hipsparseSetPointerMode(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                  HIPSPARSE_POINTER_MODE_HOST);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  stat_t = hipsparseXcsrgemmNnz(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               m, n, k,
                               cast_mat_A->mat_descr_, cast_mat_A->get_nnz(),
                               cast_mat_A->mat_.row_offset, cast_mat_A->mat_.col,
                               cast_mat_B->mat_descr_, cast_mat_B->get_nnz(),
                               cast_mat_B->mat_.row_offset, cast_mat_B->mat_.col,
                               this->mat_descr_, this->mat_.row_offset,
                               &nnzC);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  allocate_gpu(nnzC, &this->mat_.col);
  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  allocate_gpu(nnzC, &this->mat_.val);
  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  this->nrow_ = m;
  this->ncol_ = n;
  this->nnz_  = nnzC;

  stat_t = __cusparseXcsrgemm__(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                m, n, k,
                                // A
                                cast_mat_A->mat_descr_, cast_mat_A->get_nnz(),
                                cast_mat_A->mat_.val,
                                cast_mat_A->mat_.row_offset, cast_mat_A->mat_.col,
                                // B
                                cast_mat_B->mat_descr_, cast_mat_B->get_nnz(),
                                cast_mat_B->mat_.val,
                                cast_mat_B->mat_.row_offset, cast_mat_B->mat_.col,
                                // C
                                this->mat_descr_,
                                this->mat_.val,
                                this->mat_.row_offset, this->mat_.col);
  CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::Gershgorin(ValueType &lambda_min,
                                                    ValueType &lambda_max) const {
  return false;
}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::MatrixAdd(const BaseMatrix<ValueType> &mat, const ValueType alpha,
                                                   const ValueType beta, const bool structure) {

  if (this->get_nnz() > 0) {

    const GPUAcceleratorMatrixCSR<ValueType> *cast_mat = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat);
    assert(cast_mat != NULL);

    assert(cast_mat->get_nrow() == this->get_nrow());
    assert(cast_mat->get_ncol() == this->get_ncol());
    assert(this    ->get_nnz() > 0);  
    assert(cast_mat->get_nnz() > 0);

    if (structure == false) {

      int nrow = this->get_nrow();
      dim3 BlockSize(this->local_backend_.GPU_block_size);
      dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

      kernel_csr_add_csr_same_struct<ValueType, int> <<<GridSize, BlockSize>>> (nrow,
                                                                                this->mat_.row_offset, this->mat_.col,
                                                                                cast_mat->mat_.row_offset,
                                                                                cast_mat->mat_.col, cast_mat->mat_.val,
                                                                                alpha, beta, this->mat_.val);

      CHECK_CUDA_ERROR(__FILE__, __LINE__);

    } else {
      // New structure with CUSPARSE routines

      int m = this->get_nrow();
      int n = this->get_ncol();
      int *csrRowPtrC = NULL;
      int *csrColC = NULL;
      ValueType *csrValC = NULL;
      int nnzC;

      allocate_gpu(m+1, &csrRowPtrC);

      hipsparseStatus_t stat_t;

      hipsparseMatDescr_t desc_mat_C = 0;

      stat_t = hipsparseCreateMatDescr(&desc_mat_C);
      CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

      stat_t = hipsparseSetMatIndexBase(desc_mat_C, HIPSPARSE_INDEX_BASE_ZERO);
      CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

      stat_t = hipsparseSetMatType(desc_mat_C, HIPSPARSE_MATRIX_TYPE_GENERAL);
      CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

      stat_t = hipsparseSetPointerMode(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                      HIPSPARSE_POINTER_MODE_HOST);
      CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

      stat_t = hipsparseXcsrgeamNnz(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                   m, n,
                                   this->mat_descr_, this->get_nnz(),
                                   this->mat_.row_offset, this->mat_.col,
                                   cast_mat->mat_descr_, cast_mat->get_nnz(),
                                   cast_mat->mat_.row_offset, cast_mat->mat_.col,
                                   desc_mat_C, csrRowPtrC,
                                   &nnzC);
      CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

      allocate_gpu(nnzC, &csrColC);
      allocate_gpu(nnzC, &csrValC);

      stat_t = __cusparseXcsrgeam__(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                                    m, n,
                                    // A
                                    &alpha,
                                    this->mat_descr_, this->get_nnz(),
                                    this->mat_.val,
                                    this->mat_.row_offset, this->mat_.col,
                                    // B
                                    &beta,
                                    cast_mat->mat_descr_, cast_mat->get_nnz(),
                                    cast_mat->mat_.val,
                                    cast_mat->mat_.row_offset, cast_mat->mat_.col,
                                    // C
                                    desc_mat_C,
                                    csrValC,
                                    csrRowPtrC, csrColC);

      CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

      stat_t = hipsparseDestroyMatDescr(desc_mat_C);
      CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

      this->Clear();

      this->mat_.row_offset = csrRowPtrC;
      this->mat_.col = csrColC;
      this->mat_.val = csrValC;

      this->nrow_ = m;
      this->ncol_ = n;
      this->nnz_  = nnzC;

    }

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::Compress(const double drop_off) {

  if (this->get_nnz() > 0) {

    GPUAcceleratorMatrixCSR<ValueType> tmp(this->local_backend_);

    tmp.CopyFrom(*this);

    int mat_nnz = 0;

    int *row_offset = NULL;
    allocate_gpu(this->get_nrow()+1, &row_offset);

    int *mat_row_offset = NULL;
    allocate_gpu(this->get_nrow()+1, &mat_row_offset);
    
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    this->get_nrow()+1, row_offset); 


    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(this->get_nrow() / this->local_backend_.GPU_block_size + 1);
    
    kernel_csr_compress_count_nrow<ValueType, int> <<<GridSize, BlockSize>>> (this->mat_.row_offset,
                                                                              this->mat_.col,
                                                                              this->mat_.val,
                                                                              this->get_nrow(),
                                                                              drop_off,
                                                                              row_offset);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      

    // TODO
    cum_sum<int, 256>(mat_row_offset, row_offset, this->get_nrow());
  
    // get the new mat nnz
    hipMemcpy(&mat_nnz, &mat_row_offset[this->get_nrow()],
               sizeof(int), hipMemcpyDeviceToHost);
    
    this->AllocateCSR(mat_nnz, this->get_nrow(), this->get_ncol());

    // TODO - just exchange memory pointers
    // copy row_offset
    hipMemcpy(this->mat_.row_offset, mat_row_offset,
               (this->get_nrow()+1)*sizeof(int), hipMemcpyDeviceToDevice);
    
    
    // copy col and val

    kernel_csr_compress_copy<ValueType, int> <<<GridSize, BlockSize>>> (tmp.mat_.row_offset,
                                                                        tmp.mat_.col,
                                                                        tmp.mat_.val,
                                                                        tmp.get_nrow(),
                                                                        drop_off,
                                                                        this->mat_.row_offset,
                                                                        this->mat_.col,
                                                                        this->mat_.val);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      


    free_gpu(&row_offset);
    free_gpu(&mat_row_offset);
        
    
  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<double>::Transpose(void) {

  if (this->get_nnz() > 0) {

    GPUAcceleratorMatrixCSR<double> tmp(this->local_backend_);

    tmp.CopyFrom(*this);

    this->Clear();
    this->AllocateCSR(tmp.get_nnz(), tmp.get_ncol(), tmp.get_nrow());

    hipsparseStatus_t stat_t;

    stat_t = hipsparseDcsr2csc(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                              tmp.get_nrow(), tmp.get_ncol(), tmp.get_nnz(),
                              tmp.mat_.val, tmp.mat_.row_offset, tmp.mat_.col,
                              this->mat_.val, this->mat_.col, this->mat_.row_offset,
                              HIPSPARSE_ACTION_NUMERIC,
                              HIPSPARSE_INDEX_BASE_ZERO);

    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixCSR<float>::Transpose(void) {

  if (this->get_nnz() > 0) {

    GPUAcceleratorMatrixCSR<float> tmp(this->local_backend_);

    tmp.CopyFrom(*this);

    this->Clear();
    this->AllocateCSR(tmp.get_nnz(), tmp.get_ncol(), tmp.get_nrow());

    hipsparseStatus_t stat_t;

    stat_t = hipsparseScsr2csc(CUSPARSE_HANDLE(this->local_backend_.GPU_cusparse_handle),
                              tmp.get_nrow(), tmp.get_ncol(), tmp.get_nnz(),
                              tmp.mat_.val, tmp.mat_.row_offset, tmp.mat_.col,
                              this->mat_.val, this->mat_.col, this->mat_.row_offset,
                              HIPSPARSE_ACTION_NUMERIC,
                              HIPSPARSE_INDEX_BASE_ZERO);

    CHECK_CUSPARSE_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ReplaceColumnVector(const int idx, const BaseVector<ValueType> &vec) {

  assert(&vec != NULL);
  assert(vec.get_size() == this->nrow_);

  if (this->get_nnz() > 0) {

    const GPUAcceleratorVector<ValueType> *cast_vec = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&vec);
    assert(cast_vec != NULL);

    int *row_offset = NULL;
    int *col = NULL;
    ValueType *val = NULL;

    int nrow = this->get_nrow();
    int ncol = this->get_ncol();

    allocate_gpu(nrow+1, &row_offset);

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_csr_replace_column_vector_offset<ValueType, int> <<<GridSize, BlockSize>>> (this->mat_.row_offset,
                                                                                       this->mat_.col,
                                                                                       nrow,
                                                                                       idx,
                                                                                       cast_vec->vec_,
                                                                                       row_offset);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    int *host_offset = NULL;
    allocate_host(nrow+1, &host_offset);

    hipMemcpy(host_offset,
               row_offset,
               sizeof(int)*(nrow+1),
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    host_offset[0] = 0;
    for (int i=0; i<nrow; ++i)
      host_offset[i+1] += host_offset[i];

    int nnz  = host_offset[nrow];

    hipMemcpy(row_offset,
               host_offset,
               sizeof(int)*(nrow+1),
               hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    allocate_gpu(nnz, &col);
    allocate_gpu(nnz, &val);

    kernel_csr_replace_column_vector<ValueType, int> <<<GridSize, BlockSize>>> (this->mat_.row_offset,
                                                                                this->mat_.col,
                                                                                this->mat_.val,
                                                                                nrow,
                                                                                idx,
                                                                                cast_vec->vec_,
                                                                                row_offset,
                                                                                col,
                                                                                val);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    this->Clear();
    this->SetDataPtrCSR(&row_offset, &col, &val, nnz, nrow, ncol);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ExtractColumnVector(const int idx, BaseVector<ValueType> *vec) const {

  assert(vec != NULL);
  assert(vec->get_size() == this->nrow_);

  if (this->get_nnz() > 0) {

    GPUAcceleratorVector<ValueType> *cast_vec = dynamic_cast<GPUAcceleratorVector<ValueType>*> (vec);
    assert(cast_vec != NULL);

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(this->get_nrow() / this->local_backend_.GPU_block_size + 1);

    kernel_csr_extract_column_vector<ValueType, int> <<<GridSize, BlockSize>>> (this->mat_.row_offset,
                                                                                this->mat_.col,
                                                                                this->mat_.val,
                                                                                this->get_nrow(),
                                                                                idx,
                                                                                cast_vec->vec_);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCSR<ValueType>::ExtractRowVector(const int idx, BaseVector<ValueType> *vec) const {

  assert(vec != NULL);
  assert(vec->get_size() == this->ncol_);

  if (this->get_nnz() > 0) {

    GPUAcceleratorVector<ValueType> *cast_vec = dynamic_cast<GPUAcceleratorVector<ValueType>*> (vec);
    assert(cast_vec != NULL);

    cast_vec->Zeros();

    // Get nnz of row idx
    int nnz[2];

    hipMemcpy(nnz,
               this->mat_.row_offset+idx,
               2*sizeof(int),
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    int row_nnz = nnz[1] - nnz[0];

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(row_nnz / this->local_backend_.GPU_block_size + 1);

    kernel_csr_extract_row_vector<ValueType, int> <<<GridSize, BlockSize>>> (this->mat_.row_offset,
                                                                             this->mat_.col,
                                                                             this->mat_.val,
                                                                             row_nnz,
                                                                             idx,
                                                                             cast_vec->vec_);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}


template class GPUAcceleratorMatrixCSR<double>;
template class GPUAcceleratorMatrixCSR<float>;

}
