#include "hip/hip_runtime.h"
// **************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2015  PARALUTION Labs UG (haftungsbeschränkt) & Co. KG
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRA 706051
//                        Vertreten durch:
//                        PARALUTION Labs Verwaltungs UG (haftungsbeschränkt)
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRB 721277
//                        Geschäftsführer: Dimitar Lukarski, Nico Trost
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// **************************************************************************



// PARALUTION version 1.0.0 


#include "../../utils/def.hpp"
#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_coo.hpp"
#include "gpu_matrix_dia.hpp"
#include "gpu_matrix_ell.hpp"
#include "gpu_matrix_hyb.hpp"
#include "gpu_matrix_mcsr.hpp"
#include "gpu_matrix_bcsr.hpp"
#include "gpu_matrix_dense.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_dense.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_dense.hpp"
#include "gpu_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <hip/hip_runtime.h>

namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixDENSE<ValueType>::GPUAcceleratorMatrixDENSE() {

  // no default constructors
  LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixDENSE<ValueType>::GPUAcceleratorMatrixDENSE(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "GPUAcceleratorMatrixDENSE::GPUAcceleratorMatrixDENSE()",
            "constructor with local_backend");

  this->mat_.val = NULL;
  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

}


template <typename ValueType>
GPUAcceleratorMatrixDENSE<ValueType>::~GPUAcceleratorMatrixDENSE() {

  LOG_DEBUG(this, "GPUAcceleratorMatrixDENSE::~GPUAcceleratorMatrixDENSE()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixDENSE<ValueType>");

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::AllocateDENSE(const int nrow, const int ncol) {

  assert( ncol  >= 0);
  assert( nrow  >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nrow*ncol > 0) {

    allocate_gpu(nrow*ncol, &this->mat_.val);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nrow*ncol, mat_.val);   

    this->nrow_ = nrow;
    this->ncol_ = ncol;
    this->nnz_  = nrow*ncol;

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    free_gpu(&this->mat_.val);

    this->nrow_ = 0;
    this->ncol_ = 0;
    this->nnz_  = 0;

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::SetDataPtrDENSE(ValueType **val, const int nrow, const int ncol) {

  assert(*val != NULL);
  assert(nrow > 0);
  assert(ncol > 0);

  this->Clear();

  hipDeviceSynchronize();

  this->nrow_ = nrow;
  this->ncol_ = ncol;
  this->nnz_  = nrow*ncol;

  this->mat_.val = *val;

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::LeaveDataPtrDENSE(ValueType **val) {

  assert(this->nrow_ > 0);
  assert(this->ncol_ > 0);
  assert(this->nnz_ > 0);
  assert(this->nnz_  == this->nrow_*this->ncol_);

  hipDeviceSynchronize();

  *val = this->mat_.val;

  this->mat_.val = NULL;

  this->nrow_ = 0;
  this->ncol_ = 0;
  this->nnz_  = 0;

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.val,     // dst
                 cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixDENSE<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateDENSE(this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {
      
      hipMemcpy(cast_mat->mat_.val, // dst
                 this->mat_.val,     // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) { 

        hipMemcpy(this->mat_.val,         // dst
                   gpu_cast_mat->mat_.val, // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToDevice);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixDENSE<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateDENSE(dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {

        hipMemcpy(gpu_cast_mat->mat_.val, // dst
                   this->mat_.val,         // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToHost);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFromHostAsync(const HostMatrix<ValueType> &src) {

  const HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpyAsync(this->mat_.val,     // dst
                      cast_mat->mat_.val, // src
                      this->get_nnz()*sizeof(ValueType), // size
                      hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyToHostAsync(HostMatrix<ValueType> *dst) const {

  HostMatrixDENSE<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixDENSE<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateDENSE(this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {
      
      hipMemcpyAsync(cast_mat->mat_.val, // dst
                      this->mat_.val,     // src
                      this->get_nnz()*sizeof(ValueType), // size
                      hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyFromAsync(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixDENSE<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateDENSE(src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) { 

      hipMemcpy(this->mat_.val,         // dst
                 gpu_cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHostAsync(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixDENSE<ValueType>::CopyToAsync(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixDENSE<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixDENSE<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateDENSE(dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {

        hipMemcpy(gpu_cast_mat->mat_.val, // dst
                   this->mat_.val,         // src
                   this->get_nnz()*sizeof(ValueType), // size
                   hipMemcpyDeviceToHost);    
        CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHostAsync(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
bool GPUAcceleratorMatrixDENSE<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;

  const GPUAcceleratorMatrixDENSE<ValueType>   *cast_mat_dense;
  
  if ((cast_mat_dense = dynamic_cast<const GPUAcceleratorMatrixDENSE<ValueType>*> (&mat)) != NULL) {

      this->CopyFrom(*cast_mat_dense);
      return true;

  }

  /*
  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {
    
    this->Clear();
    
    FATAL_ERROR(__FILE__, __LINE__);
    
    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = cast_mat_csr->get_nnz();
    
    return true;
    
  }
  */

  return false;

}

template <>
void GPUAcceleratorMatrixDENSE<double>::Apply(const BaseVector<double> &in, BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in);
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipblasStatus_t stat_t;

    const double alpha = double(1.0);
    const double beta  = double(0.0);

    if (DENSE_IND_BASE == 0) {

      stat_t = hipblasDgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_N,
                           this->get_nrow(), this->get_ncol(),
                           &alpha,
                           this->mat_.val, this->get_nrow(),
                           cast_in->vec_, 1,
                           &beta, cast_out->vec_, 1);

      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    } else {

      stat_t = hipblasDgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_T,
                           this->get_ncol(), this->get_nrow(),
                           &alpha,
                           this->mat_.val, this->get_ncol(),
                           cast_in->vec_, 1,
                           &beta, cast_out->vec_, 1);

      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    }

  }

}

template <>
void GPUAcceleratorMatrixDENSE<float>::Apply(const BaseVector<float> &in, BaseVector<float> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in);
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipblasStatus_t stat_t;

    const float alpha = float(1.0);
    const float beta  = float(0.0);

    if (DENSE_IND_BASE == 0) {

      stat_t = hipblasSgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_N,
                           this->get_nrow(), this->get_ncol(),
                           &alpha,
                           this->mat_.val, this->get_nrow(),
                           cast_in->vec_, 1,
                           &beta, cast_out->vec_, 1);

      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    } else {

      stat_t = hipblasSgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_T,
                           this->get_ncol(), this->get_nrow(),
                           &alpha,
                           this->mat_.val, this->get_ncol(),
                           cast_in->vec_, 1,
                           &beta, cast_out->vec_, 1);

      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    }

  }

}

template <>
void GPUAcceleratorMatrixDENSE<double>::ApplyAdd(const BaseVector<double> &in, const double scalar,
                                                 BaseVector<double> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<double> *cast_in = dynamic_cast<const GPUAcceleratorVector<double>*> (&in);
    GPUAcceleratorVector<double> *cast_out      = dynamic_cast<      GPUAcceleratorVector<double>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipblasStatus_t stat_t;

    const double alpha = scalar;
    const double beta  = double(0.0);

    if (DENSE_IND_BASE == 0) {

      stat_t = hipblasDgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_N,
                           this->get_nrow(), this->get_ncol(),
                           &alpha,
                           this->mat_.val, this->get_nrow(),
                           cast_in->vec_, 1,
                           &beta, cast_out->vec_, 1);

      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    } else {

      stat_t = hipblasDgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_T,
                           this->get_ncol(), this->get_nrow(),
                           &alpha,
                           this->mat_.val, this->get_ncol(),
                           cast_in->vec_, 1,
                           &beta, cast_out->vec_, 1);

      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    }

  }

}

template <>
void GPUAcceleratorMatrixDENSE<float>::ApplyAdd(const BaseVector<float> &in, const float scalar,
                                                BaseVector<float> *out) const {
FATAL_ERROR(__FILE__, __LINE__);
  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<float> *cast_in = dynamic_cast<const GPUAcceleratorVector<float>*> (&in);
    GPUAcceleratorVector<float> *cast_out      = dynamic_cast<      GPUAcceleratorVector<float>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    hipblasStatus_t stat_t;

    const float alpha = scalar;
    const float beta  = float(0.0);

    if (DENSE_IND_BASE == 0) {

      stat_t = hipblasSgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_N,
                           this->get_nrow(), this->get_ncol(),
                           &alpha,
                           this->mat_.val, this->get_nrow(),
                           cast_in->vec_, 1,
                           &beta, cast_out->vec_, 1);

      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    } else {

      stat_t = hipblasSgemv(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_T,
                           this->get_ncol(), this->get_nrow(),
                           &alpha,
                           this->mat_.val, this->get_ncol(),
                           cast_in->vec_, 1,
                           &beta, cast_out->vec_, 1);

      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    }

  }

}

template <>
bool GPUAcceleratorMatrixDENSE<float>::MatMatMult(const BaseMatrix<float> &A, const BaseMatrix<float> &B) {

  assert((this != &A) && (this != &B));
  assert(&A != NULL);
  assert(&B != NULL);

  const GPUAcceleratorMatrixDENSE<float> *cast_mat_A = dynamic_cast<const GPUAcceleratorMatrixDENSE<float>*> (&A);
  const GPUAcceleratorMatrixDENSE<float> *cast_mat_B = dynamic_cast<const GPUAcceleratorMatrixDENSE<float>*> (&B);

  assert(cast_mat_A != NULL);
  assert(cast_mat_B != NULL);
  assert(cast_mat_A->ncol_ == cast_mat_B->nrow_);

  hipblasStatus_t stat_t;

  const float alpha = float(1.0);
  const float beta  = float(0.0);

  if (DENSE_IND_BASE == 0) {

    stat_t = hipblasSgemm(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_N, HIPBLAS_OP_N,
                         cast_mat_A->nrow_, cast_mat_B->ncol_, cast_mat_A->ncol_,
                         &alpha, cast_mat_A->mat_.val, cast_mat_A->nrow_,
                         cast_mat_B->mat_.val, cast_mat_A->ncol_, &beta,
                         this->mat_.val, cast_mat_A->nrow_);

    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  } else {

    stat_t = hipblasSgemm(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_T, HIPBLAS_OP_T,
                         cast_mat_A->nrow_, cast_mat_B->ncol_, cast_mat_A->ncol_,
                         &alpha, cast_mat_A->mat_.val, cast_mat_A->ncol_,
                         cast_mat_B->mat_.val, cast_mat_B->ncol_, &beta,
                         this->mat_.val, cast_mat_A->nrow_);

    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <>
bool GPUAcceleratorMatrixDENSE<double>::MatMatMult(const BaseMatrix<double> &A, const BaseMatrix<double> &B) {

  assert((this != &A) && (this != &B));
  assert(&A != NULL);
  assert(&B != NULL);

  const GPUAcceleratorMatrixDENSE<double> *cast_mat_A = dynamic_cast<const GPUAcceleratorMatrixDENSE<double>*> (&A);
  const GPUAcceleratorMatrixDENSE<double> *cast_mat_B = dynamic_cast<const GPUAcceleratorMatrixDENSE<double>*> (&B);

  assert(cast_mat_A != NULL);
  assert(cast_mat_B != NULL);
  assert(cast_mat_A->ncol_ == cast_mat_B->nrow_);

  hipblasStatus_t stat_t;

  const double alpha = double(1.0);
  const double beta  = double(0.0);

  if (DENSE_IND_BASE == 0) {

    stat_t = hipblasDgemm(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_N, HIPBLAS_OP_N,
                         cast_mat_A->nrow_, cast_mat_B->ncol_, cast_mat_A->ncol_,
                         &alpha, cast_mat_A->mat_.val, cast_mat_A->nrow_,
                         cast_mat_B->mat_.val, cast_mat_A->ncol_, &beta,
                         this->mat_.val, cast_mat_A->nrow_);

    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  } else {

    stat_t = hipblasDgemm(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), HIPBLAS_OP_T, HIPBLAS_OP_T,
                         cast_mat_A->nrow_, cast_mat_B->ncol_, cast_mat_A->ncol_,
                         &alpha, cast_mat_A->mat_.val, cast_mat_A->ncol_,
                         cast_mat_B->mat_.val, cast_mat_B->ncol_, &beta,
                         this->mat_.val, cast_mat_A->nrow_);

    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixDENSE<ValueType>::ReplaceColumnVector(const int idx, const BaseVector<ValueType> &vec) {

  assert(&vec != NULL);
  assert(vec.get_size() == this->get_nrow());

  if (this->get_nnz() > 0) {

    const GPUAcceleratorVector<ValueType> *cast_vec = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&vec);
    assert(cast_vec != NULL);

    const int nrow = this->get_nrow();
    const int ncol = this->get_ncol();

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_dense_replace_column_vector<ValueType, int> <<<GridSize, BlockSize>>>(cast_vec->vec_,
                                                                                 idx,
                                                                                 nrow,
                                                                                 ncol,
                                                                                 this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__,__LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixDENSE<ValueType>::ReplaceRowVector(const int idx, const BaseVector<ValueType> &vec) {

  assert(&vec != NULL);
  assert(vec.get_size() == this->get_ncol());

  if (this->get_nnz() > 0) {

    const GPUAcceleratorVector<ValueType> *cast_vec = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&vec);
    assert(cast_vec != NULL);

    const int nrow = this->get_nrow();
    const int ncol = this->get_ncol();

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(ncol / this->local_backend_.GPU_block_size + 1);

    kernel_dense_replace_row_vector<ValueType, int> <<<GridSize, BlockSize>>>(cast_vec->vec_,
                                                                              idx,
                                                                              nrow,
                                                                              ncol,
                                                                              this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__,__LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixDENSE<ValueType>::ExtractColumnVector(const int idx, BaseVector<ValueType> *vec) const {

    assert(vec != NULL);
    assert(vec->get_size() == this->get_nrow());

  if (this->get_nnz() > 0) {

    GPUAcceleratorVector<ValueType> *cast_vec = dynamic_cast<GPUAcceleratorVector<ValueType>*> (vec);
    assert(cast_vec != NULL);

    const int nrow = this->get_nrow();
    const int ncol = this->get_ncol();

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_dense_extract_column_vector<ValueType, int> <<<GridSize, BlockSize>>>(cast_vec->vec_,
                                                                                 idx,
                                                                                 nrow,
                                                                                 ncol,
                                                                                 this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__,__LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixDENSE<ValueType>::ExtractRowVector(const int idx, BaseVector<ValueType> *vec) const {

    assert(vec != NULL);
    assert(vec->get_size() == this->get_ncol());

  if (this->get_nnz() > 0) {

    GPUAcceleratorVector<ValueType> *cast_vec = dynamic_cast<GPUAcceleratorVector<ValueType>*> (vec);
    assert(cast_vec != NULL);

    const int nrow = this->get_nrow();
    const int ncol = this->get_ncol();

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(ncol / this->local_backend_.GPU_block_size + 1);

    kernel_dense_extract_row_vector<ValueType, int> <<<GridSize, BlockSize>>>(cast_vec->vec_,
                                                                              idx,
                                                                              nrow,
                                                                              ncol,
                                                                              this->mat_.val);

    CHECK_CUDA_ERROR(__FILE__,__LINE__);

  }

  return true;

}


template class GPUAcceleratorMatrixDENSE<double>;
template class GPUAcceleratorMatrixDENSE<float>;

}
