#include "hip/hip_runtime.h"
// **************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2015  PARALUTION Labs UG (haftungsbeschränkt) & Co. KG
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRA 706051
//                        Vertreten durch:
//                        PARALUTION Labs Verwaltungs UG (haftungsbeschränkt)
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRB 721277
//                        Geschäftsführer: Dimitar Lukarski, Nico Trost
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// **************************************************************************



// PARALUTION version 1.0.0 


#include "../../utils/def.hpp"
#include "gpu_vector.hpp"
#include "../base_vector.hpp"
#include "../host/host_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "../../utils/allocate_free.hpp"
#include "../../utils/math_functions.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_vector.hpp"
#include "gpu_allocate_free.hpp"

#include <hip/hip_runtime.h>
#include <hipblas.h>

namespace paralution {

template <typename ValueType>
GPUAcceleratorVector<ValueType>::GPUAcceleratorVector() {

  // no default constructors
    LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorVector<ValueType>::GPUAcceleratorVector(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "GPUAcceleratorVector::GPUAcceleratorVector()",
            "constructor with local_backend");

  this->vec_ = NULL;
  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

}


template <typename ValueType>
GPUAcceleratorVector<ValueType>::~GPUAcceleratorVector() {

  LOG_DEBUG(this, "GPUAcceleratorVector::~GPUAcceleratorVector()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorVector<ValueType>");

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::Allocate(const int n) {

  assert(n >= 0);

  if (this->get_size() >0)
    this->Clear();

  if (n > 0) {

    allocate_gpu(n, &this->vec_);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    n, this->vec_);

    this->size_ = n;
  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::SetDataPtr(ValueType **ptr, const int size) {

  assert(*ptr != NULL);
  assert(size > 0);

  hipDeviceSynchronize();

  this->vec_ = *ptr;
  this->size_ = size;

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::LeaveDataPtr(ValueType **ptr) {

  assert(this->get_size() > 0);

  hipDeviceSynchronize();
  *ptr = this->vec_;
  this->vec_ = NULL;

  this->size_ = 0 ;

}


template <typename ValueType>
void GPUAcceleratorVector<ValueType>::Clear(void) {
  
  if (this->get_size() >0) {

    free_gpu(&this->vec_);

    this->size_ = 0 ;

  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFromHost(const HostVector<ValueType> &src) {

  // CPU to GPU copy
  const HostVector<ValueType> *cast_vec;
  if ((cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src)) != NULL) {

  if (this->get_size() == 0)
    this->Allocate(cast_vec->get_size());
    
    assert(cast_vec->get_size() == this->get_size());

    if (this->get_size() >0) {
      
      
      hipblasStatus_t stat_t;
      stat_t = hipblasSetVector(this->get_size(), sizeof(ValueType),
                               cast_vec->vec_, // src
                               1,
                               this->vec_, // dst
                               1);
      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);
    }

  } else {
    
    LOG_INFO("Error unsupported GPU vector type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}



template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyToHost(HostVector<ValueType> *dst) const {

  // GPU to CPU copy
  HostVector<ValueType> *cast_vec;
  if ((cast_vec = dynamic_cast<HostVector<ValueType>*> (dst)) != NULL) {

  if (cast_vec->get_size() == 0)
    cast_vec->Allocate(this->get_size());  
    
    assert(cast_vec->get_size() == this->get_size());

    if (this->get_size() >0) {

      hipblasStatus_t stat_t;
      stat_t = hipblasGetVector(this->get_size(), sizeof(ValueType),
                               this->vec_, // src
                               1,
                               cast_vec->vec_, // dst
                               1);
      CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);
    }

  } else {
    
    LOG_INFO("Error unsupported GPU vector type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

  
}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFromHostAsync(const HostVector<ValueType> &src) {

  // CPU to GPU copy
  const HostVector<ValueType> *cast_vec;
  if ((cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src)) != NULL) {

  if (this->get_size() == 0)
    this->Allocate(cast_vec->get_size());
    
    assert(cast_vec->get_size() == this->get_size());

    if (this->get_size() >0) {

      hipMemcpyAsync(this->vec_,     // dst
                      cast_vec->vec_, // src
                      this->get_size()*sizeof(ValueType), // size
                      hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
    }

  } else {
    
    LOG_INFO("Error unsupported GPU vector type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}



template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyToHostAsync(HostVector<ValueType> *dst) const {

  // GPU to CPU copy
  HostVector<ValueType> *cast_vec;
  if ((cast_vec = dynamic_cast<HostVector<ValueType>*> (dst)) != NULL) {

  if (cast_vec->get_size() == 0)
    cast_vec->Allocate(this->get_size());  
    
    assert(cast_vec->get_size() == this->get_size());

    if (this->get_size() >0) {

      hipMemcpyAsync(cast_vec->vec_,  // dst
                      this->vec_,      // src
                      this->get_size()*sizeof(ValueType), // size
                      hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     


    }

  } else {
    
    LOG_INFO("Error unsupported GPU vector type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

  
}


template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFrom(const BaseVector<ValueType> &src) {

  const GPUAcceleratorVector<ValueType> *gpu_cast_vec;
  const HostVector<ValueType> *host_cast_vec;


    // GPU to GPU copy
    if ((gpu_cast_vec = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&src)) != NULL) {

      if (this->get_size() == 0)
        this->Allocate(gpu_cast_vec->get_size());

      assert(gpu_cast_vec->get_size() == this->get_size());

      if (this != gpu_cast_vec)  {  

        if (this->get_size() >0) {

          hipMemcpy(this->vec_,         // dst
                     gpu_cast_vec->vec_, // src
                     this->get_size()*sizeof(ValueType), // size
                     hipMemcpyDeviceToDevice);
          CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        }

      }

    } else {
      
      //GPU to CPU copy
      if ((host_cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src)) != NULL) {
        

        this->CopyFromHost(*host_cast_vec);
        
      
      } else {

        LOG_INFO("Error unsupported GPU vector type");
        this->info();
        src.info();
        FATAL_ERROR(__FILE__, __LINE__);
        
      }
      
    }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFromAsync(const BaseVector<ValueType> &src) {

  const GPUAcceleratorVector<ValueType> *gpu_cast_vec;
  const HostVector<ValueType> *host_cast_vec;


    // GPU to GPU copy
    if ((gpu_cast_vec = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&src)) != NULL) {

      if (this->get_size() == 0)
        this->Allocate(gpu_cast_vec->get_size());

      assert(gpu_cast_vec->get_size() == this->get_size());

      if (this != gpu_cast_vec)  {  

        if (this->get_size() >0) {

          hipMemcpy(this->vec_,         // dst
                     gpu_cast_vec->vec_, // src
                     this->get_size()*sizeof(ValueType), // size
                     hipMemcpyDeviceToDevice);
          CHECK_CUDA_ERROR(__FILE__, __LINE__);     
        }

      }

    } else {
      
      //GPU to CPU copy
      if ((host_cast_vec = dynamic_cast<const HostVector<ValueType>*> (&src)) != NULL) {
        

        this->CopyFromHostAsync(*host_cast_vec);
        
      
      } else {

        LOG_INFO("Error unsupported GPU vector type");
        this->info();
        src.info();
        FATAL_ERROR(__FILE__, __LINE__);
        
      }
      
    }

}


template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFrom(const BaseVector<ValueType> &src,
                                               const int src_offset,
                                               const int dst_offset,
                                               const int size) {

  assert(&src != this);
  assert(this->get_size() > 0);
  assert(src.  get_size() > 0);
  assert(size > 0);

  assert(src_offset + size <= src.get_size());
  assert(dst_offset + size <= this->get_size());

  const GPUAcceleratorVector<ValueType> *cast_src = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&src);
  assert(cast_src != NULL);

  dim3 BlockSize(this->local_backend_.GPU_block_size);
  dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);

  kernel_copy_offset_from<ValueType, int> <<<GridSize, BlockSize>>> (size, src_offset, dst_offset,
                                                                     cast_src->vec_, this->vec_);

  CHECK_CUDA_ERROR(__FILE__, __LINE__);      

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyTo(BaseVector<ValueType> *dst) const{

  GPUAcceleratorVector<ValueType> *gpu_cast_vec;
  HostVector<ValueType> *host_cast_vec;

    // GPU to GPU copy
    if ((gpu_cast_vec = dynamic_cast<GPUAcceleratorVector<ValueType>*> (dst)) != NULL) {

      if (gpu_cast_vec->get_size() == 0)
        gpu_cast_vec->Allocate(this->get_size());

      assert(gpu_cast_vec->get_size() == this->get_size());

      if (this != gpu_cast_vec)  {  

        if (this->get_size() >0) {

          hipMemcpy(gpu_cast_vec->vec_, // dst
                     this->vec_,         // src
                     this->get_size()*sizeof(ValueType), // size
                     hipMemcpyDeviceToDevice);
          CHECK_CUDA_ERROR(__FILE__, __LINE__);      
        }
      }

    } else {
      
      //GPU to CPU copy
      if ((host_cast_vec = dynamic_cast<HostVector<ValueType>*> (dst)) != NULL) {
        

        this->CopyToHost(host_cast_vec);
        
      
      } else {

        LOG_INFO("Error unsupported GPU vector type");
        this->info();
        dst->info();
        FATAL_ERROR(__FILE__, __LINE__);
        
      }
      
    }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyToAsync(BaseVector<ValueType> *dst) const{

  GPUAcceleratorVector<ValueType> *gpu_cast_vec;
  HostVector<ValueType> *host_cast_vec;

    // GPU to GPU copy
    if ((gpu_cast_vec = dynamic_cast<GPUAcceleratorVector<ValueType>*> (dst)) != NULL) {

      if (gpu_cast_vec->get_size() == 0)
        gpu_cast_vec->Allocate(this->get_size());

      assert(gpu_cast_vec->get_size() == this->get_size());

      if (this != gpu_cast_vec)  {  

        if (this->get_size() >0) {

          hipMemcpy(gpu_cast_vec->vec_, // dst
                     this->vec_,         // src
                     this->get_size()*sizeof(ValueType), // size
                     hipMemcpyDeviceToDevice);
          CHECK_CUDA_ERROR(__FILE__, __LINE__);      
        }
      }

    } else {
      
      //GPU to CPU copy
      if ((host_cast_vec = dynamic_cast<HostVector<ValueType>*> (dst)) != NULL) {
        

        this->CopyToHostAsync(host_cast_vec);
        
      
      } else {

        LOG_INFO("Error unsupported GPU vector type");
        this->info();
        dst->info();
        FATAL_ERROR(__FILE__, __LINE__);
        
      }
      
    }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFromFloat(const BaseVector<float> &src) {

  const GPUAcceleratorVector<float> *gpu_cast_vec;
  
  // GPU to GPU copy
  if ((gpu_cast_vec = dynamic_cast<const GPUAcceleratorVector<float>*> (&src)) != NULL) {
    
    if (this->get_size() == 0)
      this->Allocate(gpu_cast_vec->get_size());
    
    assert(gpu_cast_vec->get_size() == this->get_size());
    
    if (this->get_size() > 0) {
      
      dim3 BlockSize(this->local_backend_.GPU_block_size);
      dim3 GridSize(this->get_size() / this->local_backend_.GPU_block_size + 1);
      
      kernel_copy_from_float<ValueType, int> <<<GridSize, BlockSize>>>(this->get_size(), gpu_cast_vec->vec_, this->vec_);
      
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
    
  } else {
    
    LOG_INFO("Error unsupported GPU vector type");
    FATAL_ERROR(__FILE__, __LINE__);
    
  }
  
}
  
template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFromDouble(const BaseVector<double> &src) {

  const GPUAcceleratorVector<double> *gpu_cast_vec;

  // GPU to GPU copy
  if ((gpu_cast_vec = dynamic_cast<const GPUAcceleratorVector<double>*> (&src)) != NULL) {
    
    if (this->get_size() == 0)
      this->Allocate(gpu_cast_vec->get_size());
    
    assert(gpu_cast_vec->get_size() == this->get_size());
    
    
    if (this->get_size()  >0) {

      dim3 BlockSize(this->local_backend_.GPU_block_size);
      dim3 GridSize(this->get_size() / this->local_backend_.GPU_block_size + 1);
      
      kernel_copy_from_double<ValueType, int> <<<GridSize, BlockSize>>>(this->get_size(), gpu_cast_vec->vec_, this->vec_);
      
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
    
  } else {
    LOG_INFO("Error unsupported GPU vector type");
    FATAL_ERROR(__FILE__, __LINE__);
    
  }
  
}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFromData(const ValueType *data) {

  if (this->get_size() > 0) {

    hipMemcpy(this->vec_,                         // dst
               data,                               // src
               this->get_size()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyToData(ValueType *data) const {

  if (this->get_size() > 0) {

    hipMemcpy(data,                               // dst
               this->vec_,                         // src
               this->get_size()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::Zeros(void) {

  if (this->get_size() > 0) {

    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    this->get_size(), this->vec_);
    
  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::Ones(void) {

  if (this->get_size() > 0)
    set_to_one_gpu(this->local_backend_.GPU_block_size, 
                   this->local_backend_.GPU_max_threads,
                   this->get_size(), this->vec_);

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::SetValues(const ValueType val) {

  LOG_INFO("GPUAcceleratorVector::SetValues NYI");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
void GPUAcceleratorVector<double>::AddScale(const BaseVector<double> &x, const double alpha) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());
    
    const GPUAcceleratorVector<double> *cast_x = dynamic_cast<const GPUAcceleratorVector<double>*> (&x);
    assert(cast_x != NULL);
    
    hipblasStatus_t stat_t;
    
    stat_t = hipblasDaxpy(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), 
                         this->get_size(), 
                         &alpha, 
                         cast_x->vec_, 1,
                         this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void GPUAcceleratorVector<float>::AddScale(const BaseVector<float> &x, const float alpha) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());
    
    const GPUAcceleratorVector<float> *cast_x = dynamic_cast<const GPUAcceleratorVector<float>*> (&x);
    assert(cast_x != NULL);
    
    hipblasStatus_t stat_t;
    
    stat_t = hipblasSaxpy(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle), 
                         this->get_size(), 
                         &alpha, 
                         cast_x->vec_, 1,
                         this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void GPUAcceleratorVector<int>::AddScale(const BaseVector<int> &x, const int alpha) {

  LOG_INFO("No int CUBLAS axpy function");
  FATAL_ERROR(__FILE__, __LINE__);
 
}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::ScaleAdd(const ValueType alpha, const BaseVector<ValueType> &x) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());

    const GPUAcceleratorVector<ValueType> *cast_x = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&x);
    assert(cast_x != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);

    kernel_scaleadd<ValueType, int> <<<GridSize, BlockSize>>> (size, alpha, cast_x->vec_, this->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::ScaleAddScale(const ValueType alpha, const BaseVector<ValueType> &x, const ValueType beta) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());

    const GPUAcceleratorVector<ValueType> *cast_x = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&x);
    assert(cast_x != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);

    kernel_scaleaddscale<ValueType, int> <<<GridSize, BlockSize>>> (size, alpha, beta, cast_x->vec_, this->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::ScaleAddScale(const ValueType alpha, const BaseVector<ValueType> &x, const ValueType beta,
                                          const int src_offset, const int dst_offset,const int size) {

  if (this->get_size() > 0) {

    assert(this->get_size() > 0);
    assert(x.get_size() > 0);
    assert(size > 0);
    assert(src_offset + size <= x.get_size());
    assert(dst_offset + size <= this->get_size());

    const GPUAcceleratorVector<ValueType> *cast_x = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&x);
    assert(cast_x != NULL);

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);

    kernel_scaleaddscale_offset<ValueType, int> <<<GridSize, BlockSize>>> (size, src_offset, dst_offset,
                                                                           alpha, beta, cast_x->vec_, this->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::ScaleAdd2(const ValueType alpha, const BaseVector<ValueType> &x,
                                                const ValueType beta, const BaseVector<ValueType> &y,
                                                const ValueType gamma) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());
    assert(this->get_size() == y.get_size());

    const GPUAcceleratorVector<ValueType> *cast_x = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&x);
    const GPUAcceleratorVector<ValueType> *cast_y = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&y);
    assert(cast_x != NULL);
    assert(cast_y != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);

    kernel_scaleadd2<ValueType, int> <<<GridSize, BlockSize>>> (size, alpha, beta, gamma, cast_x->vec_, cast_y->vec_, this->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <>
void GPUAcceleratorVector<double>::Scale(const double alpha) {

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDscal(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                         this->get_size(), &alpha,
                         this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void GPUAcceleratorVector<float>::Scale(const float alpha) {

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasSscal(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                         this->get_size(), &alpha,
                         this->vec_, 1);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

}

template <>
void GPUAcceleratorVector<int>::Scale(const int alpha) {

  LOG_INFO("No int CUBLAS scale function");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::PartialSum(const BaseVector<ValueType> &x) {

  LOG_INFO("GPUAcceleratorVector::PartialSum() NYI");
  FATAL_ERROR(__FILE__, __LINE__); 

}

template <>
double GPUAcceleratorVector<double>::Dot(const BaseVector<double> &x) const {

  assert(this->get_size() == x.get_size());

  const GPUAcceleratorVector<double> *cast_x = dynamic_cast<const GPUAcceleratorVector<double>*> (&x);
  assert(cast_x != NULL);

  double res = 0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDdot(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                        this->get_size(),
                        this->vec_, 1,
                        cast_x->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
float GPUAcceleratorVector<float>::Dot(const BaseVector<float> &x) const {

  assert(this->get_size() == x.get_size());

  const GPUAcceleratorVector<float> *cast_x = dynamic_cast<const GPUAcceleratorVector<float>*> (&x);
  assert(cast_x != NULL);

  float res = 0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasSdot(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                        this->get_size(),
                        this->vec_, 1,
                        cast_x->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
int GPUAcceleratorVector<int>::Dot(const BaseVector<int> &x) const {

  LOG_INFO("No int CUBLAS dot function");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
ValueType GPUAcceleratorVector<ValueType>::DotNonConj(const BaseVector<ValueType> &x) const {

  return this->Dot(x);

}

template <>
int GPUAcceleratorVector<int>::DotNonConj(const BaseVector<int> &x) const {

  LOG_INFO("No int CUBLAS dot function");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
double GPUAcceleratorVector<double>::Norm(void) const {

  double res = 0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDnrm2(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                         this->get_size(),
                         this->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
float GPUAcceleratorVector<float>::Norm(void) const {

  float res = 0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasSnrm2(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                         this->get_size(),
                         this->vec_, 1, &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
int GPUAcceleratorVector<int>::Norm(void) const {

  LOG_INFO("What is int GPUAcceleratorVector<ValueType>::Norm(void) const?");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
double GPUAcceleratorVector<double>::Reduce(void) const {

  double res = 0.0;

  if (this->get_size() > 0) {

    double *d_buffer = NULL;
    double *h_buffer = NULL;
    int GROUP_SIZE;
    int LOCAL_SIZE;
    int FinalReduceSize;

    allocate_gpu<double>(this->local_backend_.GPU_warp * 4, &d_buffer);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(this->local_backend_.GPU_warp * 4);

    GROUP_SIZE = ( size_t( ( size_t( size / ( this->local_backend_.GPU_warp * 4 ) ) + 1 ) 
                 / this->local_backend_.GPU_block_size ) + 1 ) * this->local_backend_.GPU_block_size;
    LOCAL_SIZE = GROUP_SIZE / this->local_backend_.GPU_block_size;
    

    kernel_reduce<double, int, 256> <<<GridSize, BlockSize>>> (size,
                                                               this->vec_,
                                                               d_buffer,
                                                               GROUP_SIZE,
                                                               LOCAL_SIZE);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    FinalReduceSize = this->local_backend_.GPU_warp * 4;
    allocate_host(FinalReduceSize, &h_buffer);

    hipMemcpy(h_buffer, // dst
               d_buffer, // src
               FinalReduceSize*sizeof(double), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    free_gpu<double>(&d_buffer);

    for ( int i=0; i<FinalReduceSize; ++i ) {
      res += h_buffer[i];
    }

    free_host(&h_buffer);

  }

  return res;

}

template <>
float GPUAcceleratorVector<float>::Reduce(void) const {

  float res = 0.0;

  if (this->get_size() > 0) {

    float *d_buffer = NULL;
    float *h_buffer = NULL;
    int GROUP_SIZE;
    int LOCAL_SIZE;
    int FinalReduceSize;

    allocate_gpu<float>(this->local_backend_.GPU_warp * 4, &d_buffer);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(this->local_backend_.GPU_warp * 4);

    GROUP_SIZE = ( size_t( ( size_t( size / ( this->local_backend_.GPU_warp * 4 ) ) + 1 ) 
                 / this->local_backend_.GPU_block_size ) + 1 ) * this->local_backend_.GPU_block_size;
    LOCAL_SIZE = GROUP_SIZE / this->local_backend_.GPU_block_size;
    
    kernel_reduce<float, int, 256> <<<GridSize, BlockSize>>> (size,
                                                              this->vec_,
                                                              d_buffer,
                                                              GROUP_SIZE,
                                                              LOCAL_SIZE);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    FinalReduceSize = this->local_backend_.GPU_warp * 4;
    allocate_host(FinalReduceSize, &h_buffer);

    hipMemcpy(h_buffer,         // dst
               d_buffer, // src
               FinalReduceSize*sizeof(float), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__); 

    free_gpu<float>(&d_buffer);

    for ( int i=0; i<FinalReduceSize; ++i ) {
      res += h_buffer[i];
    }

    free_host(&h_buffer);

  }

  return res;

}

template <>
int GPUAcceleratorVector<int>::Reduce(void) const {

  LOG_INFO("Reduce<int> not implemented");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
double GPUAcceleratorVector<double>::Asum(void) const {

  double res = 0.0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasDasum(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                         this->get_size(),
                         this->vec_, 1,
                         &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
float GPUAcceleratorVector<float>::Asum(void) const {

  float res = 0.0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasSasum(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                         this->get_size(),
                         this->vec_, 1,
                         &res);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

  }

  return res;

}

template <>
int GPUAcceleratorVector<int>::Asum(void) const {

  LOG_INFO("Asum<int> not implemented");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <>
int GPUAcceleratorVector<double>::Amax(double &value) const {

  int index = 0;
  value = 0.0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasIdamax(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                          this->get_size(),
                          this->vec_, 1, &index);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    // cublas returns 1-based indexing
    --index;

    hipMemcpy(&value,
               this->vec_+index,
               sizeof(double),
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  value = paralution_abs(value);
  return index;

}

template <>
int GPUAcceleratorVector<float>::Amax(float &value) const {

  int index = 0;
  value = 0.0;

  if (this->get_size() > 0) {

    hipblasStatus_t stat_t;

    stat_t = hipblasIsamax(CUBLAS_HANDLE(this->local_backend_.GPU_cublas_handle),
                          this->get_size(),
                          this->vec_, 1, &index);
    CHECK_CUBLAS_ERROR(stat_t, __FILE__, __LINE__);

    // cublas returns 1-based indexing
    --index;

    hipMemcpy(&value,
               this->vec_+index,
               sizeof(float),
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  value = paralution_abs(value);
  return index;

}

template <>
int GPUAcceleratorVector<int>::Amax(int &value) const {

  LOG_INFO("Amax<int> not implemented");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::PointWiseMult(const BaseVector<ValueType> &x) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());

    const GPUAcceleratorVector<ValueType> *cast_x = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&x);
    assert(cast_x != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);

    kernel_pointwisemult<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_x->vec_, this->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::PointWiseMult(const BaseVector<ValueType> &x, const BaseVector<ValueType> &y) {

  if (this->get_size() > 0) {

    assert(this->get_size() == x.get_size());
    assert(this->get_size() == y.get_size());

    const GPUAcceleratorVector<ValueType> *cast_x = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&x);
    const GPUAcceleratorVector<ValueType> *cast_y = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&y);
    assert(cast_x != NULL);
    assert(cast_y != NULL);

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);

    kernel_pointwisemult2<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_x->vec_, cast_y->vec_, this->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::Permute(const BaseVector<int> &permutation) {

  if (this->get_size() > 0) {

    assert(&permutation != NULL);
    assert(this->get_size() == permutation.get_size());
    
    const GPUAcceleratorVector<int> *cast_perm = dynamic_cast<const GPUAcceleratorVector<int>*> (&permutation);
    assert(cast_perm != NULL);
    
    GPUAcceleratorVector<ValueType> vec_tmp(this->local_backend_);     
    vec_tmp.Allocate(this->get_size());
    vec_tmp.CopyFrom(*this);
    
    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);
    
    //    this->vec_[ cast_perm->vec_[i] ] = vec_tmp.vec_[i];  
    kernel_permute<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_perm->vec_, vec_tmp.vec_, this->vec_);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::PermuteBackward(const BaseVector<int> &permutation) {

  if (this->get_size() > 0) {

    assert(&permutation != NULL);
    assert(this->get_size() == permutation.get_size());
    
    const GPUAcceleratorVector<int> *cast_perm = dynamic_cast<const GPUAcceleratorVector<int>*> (&permutation);
    assert(cast_perm != NULL);
    
    GPUAcceleratorVector<ValueType> vec_tmp(this->local_backend_);   
    vec_tmp.Allocate(this->get_size());
    vec_tmp.CopyFrom(*this);
    
    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);
    
    //    this->vec_[i] = vec_tmp.vec_[ cast_perm->vec_[i] ];
    kernel_permute_backward<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_perm->vec_, vec_tmp.vec_, this->vec_);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFromPermute(const BaseVector<ValueType> &src,
                                                      const BaseVector<int> &permutation) { 

  if (this->get_size() > 0) {

    assert(this != &src);
    
    const GPUAcceleratorVector<ValueType> *cast_vec = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&src);
    const GPUAcceleratorVector<int> *cast_perm      = dynamic_cast<const GPUAcceleratorVector<int>*> (&permutation) ; 
    assert(cast_perm != NULL);
    assert(cast_vec  != NULL);
    
    assert(cast_vec ->get_size() == this->get_size());
    assert(cast_perm->get_size() == this->get_size());
    
    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);
    
    //    this->vec_[ cast_perm->vec_[i] ] = cast_vec->vec_[i];
    kernel_permute<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_perm->vec_, cast_vec->vec_, this->vec_);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
  }

}

template <typename ValueType>
void GPUAcceleratorVector<ValueType>::CopyFromPermuteBackward(const BaseVector<ValueType> &src,
                                                              const BaseVector<int> &permutation) {

  if (this->get_size() > 0) {

    assert(this != &src);
    
    const GPUAcceleratorVector<ValueType> *cast_vec = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&src);
    const GPUAcceleratorVector<int> *cast_perm      = dynamic_cast<const GPUAcceleratorVector<int>*> (&permutation) ; 
    assert(cast_perm != NULL);
    assert(cast_vec  != NULL);
    
    assert(cast_vec ->get_size() == this->get_size());
    assert(cast_perm->get_size() == this->get_size());
    
    
    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);
    
    //    this->vec_[i] = cast_vec->vec_[ cast_perm->vec_[i] ];
    kernel_permute_backward<ValueType, int> <<<GridSize, BlockSize>>> (size, cast_perm->vec_, cast_vec->vec_, this->vec_);
    
    CHECK_CUDA_ERROR(__FILE__, __LINE__);      
  }

}

template <>
void GPUAcceleratorVector<double>::Power(const double power) {

  if (this->get_size() > 0) {

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);

    kernel_powerd<int> <<<GridSize, BlockSize>>> (size, power, this->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <>
void GPUAcceleratorVector<float>::Power(const double power) {

  if (this->get_size() > 0) {

    int size = this->get_size();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(size / this->local_backend_.GPU_block_size + 1);

    kernel_powerf<int> <<<GridSize, BlockSize>>> (size, power, this->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <>
void GPUAcceleratorVector<int>::Power(const double power) {

  if (this->get_size() > 0) {

    LOG_INFO("GPUAcceleratorVector::Power(), no pow() for int in CUDA");
    FATAL_ERROR(__FILE__, __LINE__);


  }

}


template class GPUAcceleratorVector<double>;
template class GPUAcceleratorVector<float>;
template class GPUAcceleratorVector<int>;

}
