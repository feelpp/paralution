#include "hip/hip_runtime.h"
// **************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2015  PARALUTION Labs UG (haftungsbeschränkt) & Co. KG
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRA 706051
//                        Vertreten durch:
//                        PARALUTION Labs Verwaltungs UG (haftungsbeschränkt)
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRB 721277
//                        Geschäftsführer: Dimitar Lukarski, Nico Trost
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// **************************************************************************



// PARALUTION version 1.0.0 


#include "../../utils/def.hpp"
#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_coo.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_coo.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_coo.hpp"
#include "gpu_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixCOO<ValueType>::GPUAcceleratorMatrixCOO() {

  // no default constructors
  LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixCOO<ValueType>::GPUAcceleratorMatrixCOO(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "GPUAcceleratorMatrixCOO::GPUAcceleratorMatrixCOO()",
            "constructor with local_backend");

  this->mat_.row = NULL;  
  this->mat_.col = NULL;  
  this->mat_.val = NULL;
  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

}


template <typename ValueType>
GPUAcceleratorMatrixCOO<ValueType>::~GPUAcceleratorMatrixCOO() {

  LOG_DEBUG(this, "GPUAcceleratorMatrixCOO::~GPUAcceleratorMatrixCOO()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixCOO<ValueType>");

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::AllocateCOO(const int nnz, const int nrow, const int ncol) {

  assert(nnz >= 0);
  assert(ncol >= 0);
  assert(nrow >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nnz > 0) {

    allocate_gpu(nnz, &this->mat_.row);
    allocate_gpu(nnz, &this->mat_.col);
    allocate_gpu(nnz, &this->mat_.val);
 
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, this->mat_.row);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, this->mat_.col);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, this->mat_.val);

    this->nrow_ = nrow;
    this->ncol_ = ncol;
    this->nnz_  = nnz;

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::SetDataPtrCOO(int **row, int **col, ValueType **val,
                                                       const int nnz, const int nrow, const int ncol) {

  assert(*row != NULL);
  assert(*col != NULL);
  assert(*val != NULL);
  assert(nnz > 0);
  assert(nrow > 0);
  assert(ncol > 0);

  this->Clear();

  this->nrow_ = nrow;
  this->ncol_ = ncol;
  this->nnz_  = nnz;

  hipDeviceSynchronize();

  this->mat_.row = *row;
  this->mat_.col = *col;
  this->mat_.val = *val;

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::LeaveDataPtrCOO(int **row, int **col, ValueType **val) {

  assert(this->get_nrow() > 0);
  assert(this->get_ncol() > 0);
  assert(this->get_nnz() > 0);

  hipDeviceSynchronize();

  // see free_host function for details
  *row = this->mat_.row;
  *col = this->mat_.col;
  *val = this->mat_.val;

  this->mat_.row = NULL;
  this->mat_.col = NULL;
  this->mat_.val = NULL;

  this->nrow_ = 0;
  this->ncol_ = 0;
  this->nnz_  = 0;

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    free_gpu(&this->mat_.row);
    free_gpu(&this->mat_.col);
    free_gpu(&this->mat_.val);

    this->nrow_ = 0;
    this->ncol_ = 0;
    this->nnz_  = 0;

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixCOO<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixCOO<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateCOO(src.get_nnz(), src.get_nrow(), src.get_ncol() );

  if (this->get_nnz() > 0) {

      assert(this->get_nnz()  == src.get_nnz());
      assert(this->get_nrow()  == src.get_nrow());
      assert(this->get_ncol()  == src.get_ncol());
      
      hipMemcpy(this->mat_.row,     // dst
                 cast_mat->mat_.row, // src
                 (this->get_nnz())*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.col,     // dst
                 cast_mat->mat_.col, // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.val,     // dst
                 cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixCOO<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixCOO<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateCOO(this->get_nnz(), this->get_nrow(), this->get_ncol() );

  if (this->get_nnz() > 0) {

      assert(this->get_nnz()  == dst->get_nnz());
      assert(this->get_nrow() == dst->get_nrow());
      assert(this->get_ncol() == dst->get_ncol());
      
      hipMemcpy(cast_mat->mat_.row, // dst
                 this->mat_.row,     // src
                 this->get_nnz()*sizeof(int), // size           
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(cast_mat->mat_.col, // dst
                 this->mat_.col,     // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(cast_mat->mat_.val, // dst
                 this->mat_.val,     // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixCOO<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixCOO<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateCOO(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.row,         // dst
                 gpu_cast_mat->mat_.row, // src
                 (this->get_nnz())*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.col,         // dst
                 gpu_cast_mat->mat_.col, // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.val,         // dst
                 gpu_cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixCOO<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixCOO<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateCOO(dst->get_nnz(), dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpy(gpu_cast_mat->mat_.row, // dst
                 this->mat_.row,         // src
                 (this->get_nnz())*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.col, // dst
                 this->mat_.col,         // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.val, // dst
                 this->mat_.val,         // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyFromHostAsync(const HostMatrix<ValueType> &src) {

  const HostMatrixCOO<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixCOO<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateCOO(src.get_nnz(), src.get_nrow(), src.get_ncol() );

  if (this->get_nnz() > 0) {

      assert(this->get_nnz()  == src.get_nnz());
      assert(this->get_nrow()  == src.get_nrow());
      assert(this->get_ncol()  == src.get_ncol());
      
      hipMemcpyAsync(this->mat_.row,     // dst
                      cast_mat->mat_.row, // src
                      (this->get_nnz())*sizeof(int), // size
                      hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(this->mat_.col,     // dst
                      cast_mat->mat_.col, // src
                      this->get_nnz()*sizeof(int), // size
                      hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(this->mat_.val,     // dst
                      cast_mat->mat_.val, // src
                      this->get_nnz()*sizeof(ValueType), // size
                      hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyToHostAsync(HostMatrix<ValueType> *dst) const {

  HostMatrixCOO<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixCOO<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateCOO(this->get_nnz(), this->get_nrow(), this->get_ncol() );

  if (this->get_nnz() > 0) {

      assert(this->get_nnz()  == dst->get_nnz());
      assert(this->get_nrow() == dst->get_nrow());
      assert(this->get_ncol() == dst->get_ncol());
      
      hipMemcpyAsync(cast_mat->mat_.row, // dst
                      this->mat_.row,     // src
                      this->get_nnz()*sizeof(int), // size           
                      hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(cast_mat->mat_.col, // dst
                      this->mat_.col,     // src
                      this->get_nnz()*sizeof(int), // size
                      hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(cast_mat->mat_.val, // dst
                      this->mat_.val,     // src
                      this->get_nnz()*sizeof(ValueType), // size
                      hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyFromAsync(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixCOO<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixCOO<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateCOO(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.row,         // dst
                 gpu_cast_mat->mat_.row, // src
                 (this->get_nnz())*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.col,         // dst
                 gpu_cast_mat->mat_.col, // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.val,         // dst
                 gpu_cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHostAsync(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyToAsync(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixCOO<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixCOO<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateCOO(dst->get_nnz(), dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpy(gpu_cast_mat->mat_.row, // dst
                 this->mat_.row,         // src
                 (this->get_nnz())*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.col, // dst
                 this->mat_.col,         // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.val, // dst
                 this->mat_.val,         // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHostAsync(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyFromCOO(const int *row, const int *col, const ValueType *val) {

  // assert CSR format
  assert(this->get_mat_format() == COO);

  if (this->get_nnz() > 0) {

    assert(this->nrow_ > 0);
    assert(this->ncol_ > 0);

    hipMemcpy(this->mat_.row,              // dst
               row,                         // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    hipMemcpy(this->mat_.col,              // dst
               col,                         // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    hipMemcpy(this->mat_.val,                    // dst
               val,                               // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::CopyToCOO(int *row, int *col, ValueType *val) const {

  // assert CSR format
  assert(this->get_mat_format() == COO);

  if (this->get_nnz() > 0) {

    assert(this->nrow_ > 0);
    assert(this->ncol_ > 0);

    hipMemcpy(row,                         // dst
               this->mat_.row,              // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    hipMemcpy(col,                         // dst
               this->mat_.col,              // src
               this->get_nnz()*sizeof(int), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    hipMemcpy(val,                               // dst
               this->mat_.val,                    // src
               this->get_nnz()*sizeof(ValueType), // size
               hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
bool GPUAcceleratorMatrixCOO<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;

  const GPUAcceleratorMatrixCOO<ValueType>   *cast_mat_coo;
  if ((cast_mat_coo = dynamic_cast<const GPUAcceleratorMatrixCOO<ValueType>*> (&mat)) != NULL) {

      this->CopyFrom(*cast_mat_coo);
      return true;

  }

  /*
  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {
    
    this->Clear();
    
    FATAL_ERROR(__FILE__, __LINE__);
    
    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = cast_mat_csr->get_nnz();
    
    return true;

  }
  */

  return false;

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::Apply(const BaseVector<ValueType> &in, BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in);
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    cast_out->Zeros();

    // do not support super small matrices
    assert(this->get_nnz() > this->local_backend_.GPU_warp); 

    // ----------------------------------------------------------
    // Modified and adapted from CUSP 0.3.1, 
    // http://code.google.com/p/cusp-library/
    // NVIDIA, APACHE LICENSE 2.0
    // ----------------------------------------------------------
    // see __spmv_coo_flat(...)
    // ----------------------------------------------------------
    // CHANGELOG
    // - adapted interface
    // ----------------------------------------------------------  

    //TODO
    //move in extra file -  max_active_blocks, warp_size, block_size

    const unsigned int BLOCK_SIZE = this->local_backend_.GPU_block_size;
    //    const unsigned int MAX_BLOCKS = this->local_backend_.GPU_max_blocks;

    const unsigned int MAX_BLOCKS = 32; //  cusp::detail::device::arch::max_active_blocks(spmv_coo_flat_kernel<IndexType, ValueType, BLOCK_SIZE, UseCache>, BLOCK_SIZE, (size_t) 0);

    const unsigned int WARPS_PER_BLOCK = BLOCK_SIZE / this->local_backend_.GPU_warp;

    const unsigned int num_units  = this->get_nnz() / this->local_backend_.GPU_warp;
    const unsigned int num_warps  = std::min(num_units, WARPS_PER_BLOCK * MAX_BLOCKS);
    const unsigned int num_blocks = (num_warps + (WARPS_PER_BLOCK-1)) / WARPS_PER_BLOCK; // (N + (granularity - 1)) / granularity
    const unsigned int num_iters  = (num_units +  (num_warps-1)) / num_warps;

    const unsigned int interval_size = this->local_backend_.GPU_warp * num_iters;

    const int tail = num_units * this->local_backend_.GPU_warp; // do the last few nonzeros separately (fewer than this->local_backend_.GPU_warp elements)

    const unsigned int active_warps = (interval_size == 0) ? 0 : ((tail + (interval_size-1))/interval_size);

    int *temp_rows = NULL;
    ValueType *temp_vals = NULL;

    allocate_gpu(active_warps, &temp_rows);
    allocate_gpu(active_warps, &temp_vals);

    //  LOG_INFO("active_warps = " << active_warps);
    //  LOG_INFO("tail =" << tail);
    //  LOG_INFO("interval_size =" << interval_size);
    //  LOG_INFO("num_iters =" << num_iters);
    //  LOG_INFO("num_blocks =" << num_blocks);
    //  LOG_INFO("num_warps =" << num_warps);
    //  LOG_INFO("num_units =" << num_units);
    //  LOG_INFO("WARPS_PER_BLOCK =" << WARPS_PER_BLOCK);
    //  LOG_INFO("MAX_BLOCKS =" << MAX_BLOCKS);
    //  LOG_INFO("BLOCK_SIZE =" << BLOCK_SIZE);
    //  LOG_INFO("WARP_SIZE =" << WARP_SIZE);
    //  LOG_INFO("WARP_SIZE =" << this->local_backend_.GPU_warp);

    // TODO
    // BLOCK_SIZE == 256
    // WARP_SIZE == 32
    kernel_spmv_coo_flat<int, ValueType, 256, 32> <<<num_blocks, BLOCK_SIZE>>> (tail, interval_size,
                                                                                this->mat_.row, this->mat_.col,
                                                                                this->mat_.val,
                                                                                ValueType(1.0),
                                                                                cast_in->vec_, cast_out->vec_,
                                                                                temp_rows, temp_vals);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    // TODO
    // BLOCK_SIZE == 256
    kernel_spmv_coo_reduce_update<int, ValueType, 256> <<<1, BLOCK_SIZE>>> (active_warps, temp_rows, temp_vals,
                                                                            cast_out->vec_);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    kernel_spmv_coo_serial<int, ValueType> <<<1,1>>> (this->get_nnz() - tail,
                                                      this->mat_.row + tail,
                                                      this->mat_.col + tail,
                                                      this->mat_.val + tail,
                                                      ValueType(1.0),
                                                      cast_in->vec_, cast_out->vec_);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    free_gpu(&temp_rows);
    free_gpu(&temp_vals);

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixCOO<ValueType>::ApplyAdd(const BaseVector<ValueType> &in, const ValueType scalar,
                                                  BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in);
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    // do not support super small matrices
    assert(this->get_nnz() > this->local_backend_.GPU_warp);

    // ----------------------------------------------------------
    // Modified and adapted from CUSP 0.3.1,
    // http://code.google.com/p/cusp-library/
    // NVIDIA, APACHE LICENSE 2.0
    // ----------------------------------------------------------
    // see __spmv_coo_flat(...)
    // ----------------------------------------------------------
    // CHANGELOG
    // - adapted interface
    // ----------------------------------------------------------  

    const unsigned int BLOCK_SIZE = this->local_backend_.GPU_block_size;
    //    const unsigned int MAX_BLOCKS = this->local_backend_.GPU_max_blocks;

    const unsigned int MAX_BLOCKS = 32; //  cusp::detail::device::arch::max_active_blocks(spmv_coo_flat_kernel<IndexType, ValueType, BLOCK_SIZE, UseCache>, BLOCK_SIZE, (size_t) 0);

    const unsigned int WARPS_PER_BLOCK = BLOCK_SIZE / this->local_backend_.GPU_warp;

    const unsigned int num_units  = this->get_nnz() / this->local_backend_.GPU_warp;
    const unsigned int num_warps  = std::min(num_units, WARPS_PER_BLOCK * MAX_BLOCKS);
    const unsigned int num_blocks = (num_warps + (WARPS_PER_BLOCK-1)) / WARPS_PER_BLOCK; // (N + (granularity - 1)) / granularity
    const unsigned int num_iters  = (num_units +  (num_warps-1)) / num_warps;

    const unsigned int interval_size = this->local_backend_.GPU_warp * num_iters;

    const int tail = num_units * this->local_backend_.GPU_warp; // do the last few nonzeros separately (fewer than this->local_backend_.GPU_warp elements)

    const unsigned int active_warps = (interval_size == 0) ? 0 : ((tail + (interval_size-1))/interval_size);

    int *temp_rows = NULL;
    ValueType *temp_vals = NULL;

    allocate_gpu(active_warps, &temp_rows);
    allocate_gpu(active_warps, &temp_vals);

    // TODO
    // BLOCK_SIZE == 256
    // WARP_SIZE == 32
    kernel_spmv_coo_flat<int, ValueType, 256, 32> <<<num_blocks, BLOCK_SIZE>>> (tail, interval_size,
                                                                                this->mat_.row, this->mat_.col,
                                                                                this->mat_.val,
                                                                                scalar,
                                                                                cast_in->vec_, cast_out->vec_,
                                                                                temp_rows, temp_vals);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    // TODO
    // BLOCK_SIZE == 256
    kernel_spmv_coo_reduce_update<int, ValueType, 256> <<<1, BLOCK_SIZE>>> (active_warps, temp_rows, temp_vals,
                                                                            cast_out->vec_);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    kernel_spmv_coo_serial<int, ValueType> <<<1,1>>> (this->get_nnz() - tail,
                                                      this->mat_.row + tail,
                                                      this->mat_.col + tail,
                                                      this->mat_.val + tail,
                                                      scalar,
                                                      cast_in->vec_, cast_out->vec_);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    free_gpu(&temp_rows);
    free_gpu(&temp_vals);

  }

}

template <typename ValueType>
bool GPUAcceleratorMatrixCOO<ValueType>::Permute(const BaseVector<int> &permutation) {

  assert(&permutation != NULL);

  // symmetric permutation only
  assert(permutation.get_size() == this->get_nrow());
  assert(permutation.get_size() == this->get_ncol());

  if (this->get_nnz() > 0) {

    const GPUAcceleratorVector<int> *cast_perm = dynamic_cast<const GPUAcceleratorVector<int>*> (&permutation);
    assert(cast_perm != NULL);

    GPUAcceleratorMatrixCOO<ValueType> src(this->local_backend_);
    src.AllocateCOO(this->get_nnz(), this->get_nrow(), this->get_ncol());
    src.CopyFrom(*this);

    int nnz = this->get_nnz();
    int s = nnz;
    int k = (nnz/this->local_backend_.GPU_block_size)/this->local_backend_.GPU_max_threads + 1;
    if (k > 1) s = nnz / k;

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(s / this->local_backend_.GPU_block_size + 1);

    kernel_coo_permute<ValueType, int> <<<GridSize, BlockSize>>> (nnz,
                                                                  src.mat_.row, src.mat_.col,
                                                                  cast_perm->vec_,
                                                                  this->mat_.row, this->mat_.col);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

  return true;

}

template <typename ValueType>
bool GPUAcceleratorMatrixCOO<ValueType>::PermuteBackward(const BaseVector<int> &permutation) {

  assert(&permutation != NULL);

  // symmetric permutation only
  assert(permutation.get_size() == this->get_nrow());
  assert(permutation.get_size() == this->get_ncol());

  if (this->get_nnz() > 0) {

    const GPUAcceleratorVector<int> *cast_perm = dynamic_cast<const GPUAcceleratorVector<int>*> (&permutation);
    assert(cast_perm != NULL);

    int *pb = NULL;
    allocate_gpu(this->get_nrow(), &pb);

    int n = this->get_nrow();
    dim3 BlockSize1(this->local_backend_.GPU_block_size);
    dim3 GridSize1(n / this->local_backend_.GPU_block_size + 1);

    kernel_reverse_index<int> <<<GridSize1, BlockSize1>>> (n,
                                                           cast_perm->vec_,
                                                           pb);

    GPUAcceleratorMatrixCOO<ValueType> src(this->local_backend_);
    src.AllocateCOO(this->get_nnz(), this->get_nrow(), this->get_ncol());
    src.CopyFrom(*this);

    int nnz = this->get_nnz();
    int s = nnz;
    int k = (nnz/this->local_backend_.GPU_block_size)/this->local_backend_.GPU_max_threads + 1;
    if (k > 1) s = nnz / k;

    dim3 BlockSize2(this->local_backend_.GPU_block_size);
    dim3 GridSize2(s / this->local_backend_.GPU_block_size + 1);

    kernel_coo_permute<ValueType, int> <<<GridSize2, BlockSize2>>> (nnz,
                                                                    src.mat_.row, src.mat_.col,
                                                                    pb,
                                                                    this->mat_.row, this->mat_.col);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    free_gpu(&pb);

  }

  return true;

}


template class GPUAcceleratorMatrixCOO<double>;
template class GPUAcceleratorMatrixCOO<float>;

}
