#include "hip/hip_runtime.h"
// **************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2015  PARALUTION Labs UG (haftungsbeschränkt) & Co. KG
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRA 706051
//                        Vertreten durch:
//                        PARALUTION Labs Verwaltungs UG (haftungsbeschränkt)
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRB 721277
//                        Geschäftsführer: Dimitar Lukarski, Nico Trost
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// **************************************************************************



// PARALUTION version 1.0.0 


#include "../../utils/def.hpp"
#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_bcsr.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_bcsr.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_bcsr.hpp"
#include "gpu_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <hip/hip_runtime.h>
#include <hipsparse.h>

namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixBCSR<ValueType>::GPUAcceleratorMatrixBCSR() {

  // no default constructors
  LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixBCSR<ValueType>::GPUAcceleratorMatrixBCSR(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "GPUAcceleratorMatrixBCSR::GPUAcceleratorMatrixBCSR()",
            "constructor with local_backend");

  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

  // this is not working anyway...
  FATAL_ERROR(__FILE__, __LINE__);
}


template <typename ValueType>
GPUAcceleratorMatrixBCSR<ValueType>::~GPUAcceleratorMatrixBCSR() {

  LOG_DEBUG(this, "GPUAcceleratorMatrixBCSR::~GPUAcceleratorMatrixBCSR()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorMatrixBCSR<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixBCSR<ValueType>");

}

template <typename ValueType>
void GPUAcceleratorMatrixBCSR<ValueType>::AllocateBCSR(const int nnz, const int nrow, const int ncol) {

  assert(nnz >= 0);
  assert(ncol >= 0);
  assert(nrow >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nnz > 0) {

    FATAL_ERROR(__FILE__, __LINE__);
   

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixBCSR<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    FATAL_ERROR(__FILE__, __LINE__);


  }


}

template <typename ValueType>
void GPUAcceleratorMatrixBCSR<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixBCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixBCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateBCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    cast_mat->get_nnz();

    FATAL_ERROR(__FILE__, __LINE__);    
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixBCSR<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixBCSR<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixBCSR<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateBCSR(this->get_nnz(), this->get_nrow(), this->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    FATAL_ERROR(__FILE__, __LINE__);    
   
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixBCSR<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixBCSR<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixBCSR<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateBCSR(src.get_nnz(), src.get_nrow(), src.get_ncol() );  

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    gpu_cast_mat->get_nnz();

    FATAL_ERROR(__FILE__, __LINE__);    

    
  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixBCSR<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixBCSR<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixBCSR<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateBCSR(dst->get_nnz(), dst->get_nrow(), dst->get_ncol() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    FATAL_ERROR(__FILE__, __LINE__);    
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
bool GPUAcceleratorMatrixBCSR<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;


  const GPUAcceleratorMatrixBCSR<ValueType>   *cast_mat_bcsr;
  if ((cast_mat_bcsr = dynamic_cast<const GPUAcceleratorMatrixBCSR<ValueType>*> (&mat)) != NULL) {

      this->CopyFrom(*cast_mat_bcsr);
      return true;

  }

  /*
  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {
    
    this->Clear();
    
    FATAL_ERROR(__FILE__, __LINE__);
    
    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = cast_mat_csr->get_nnz();
    
    return true;

  }
  */


  return false;

}

template <typename ValueType>
void GPUAcceleratorMatrixBCSR<ValueType>::Apply(const BaseVector<ValueType> &in, BaseVector<ValueType> *out) const {
/*
  assert(in.  get_size() >= 0);
  assert(out->get_size() >= 0);
  assert(in.  get_size() == this->get_ncol());
  assert(out->get_size() == this->get_nrow());


  const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in) ; 
  GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out) ; 

  assert(cast_in != NULL);
  assert(cast_out!= NULL);
*/
  FATAL_ERROR(__FILE__, __LINE__);    

}


template <typename ValueType>
void GPUAcceleratorMatrixBCSR<ValueType>::ApplyAdd(const BaseVector<ValueType> &in, const ValueType scalar,
                                                  BaseVector<ValueType> *out) const {
  FATAL_ERROR(__FILE__, __LINE__);
}


template class GPUAcceleratorMatrixBCSR<double>;
template class GPUAcceleratorMatrixBCSR<float>;

}
