#include "hip/hip_runtime.h"
// **************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2015  PARALUTION Labs UG (haftungsbeschränkt) & Co. KG
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRA 706051
//                        Vertreten durch:
//                        PARALUTION Labs Verwaltungs UG (haftungsbeschränkt)
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRB 721277
//                        Geschäftsführer: Dimitar Lukarski, Nico Trost
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// **************************************************************************



// PARALUTION version 1.0.0 


#include "../../utils/def.hpp"
#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_coo.hpp"
#include "gpu_matrix_ell.hpp"
#include "gpu_matrix_hyb.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_hyb.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "../../utils/allocate_free.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_hyb.hpp"
#include "cuda_kernels_vector.hpp"
#include "gpu_allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixHYB<ValueType>::GPUAcceleratorMatrixHYB() {

  // no default constructors
  LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixHYB<ValueType>::GPUAcceleratorMatrixHYB(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "GPUAcceleratorMatrixHYB::GPUAcceleratorMatrixHYB()",
            "constructor with local_backend");

  this->mat_.ELL.val = NULL;
  this->mat_.ELL.col = NULL;
  this->mat_.ELL.max_row = 0;

  this->mat_.COO.row = NULL;  
  this->mat_.COO.col = NULL;  
  this->mat_.COO.val = NULL;

  this->ell_nnz_ = 0;
  this->coo_nnz_ = 0;

  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

}


template <typename ValueType>
GPUAcceleratorMatrixHYB<ValueType>::~GPUAcceleratorMatrixHYB() {

  LOG_DEBUG(this, "GPUAcceleratorMatrixHYB::~GPUAcceleratorMatrixHYB()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixHYB<ValueType>");

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::AllocateHYB(const int ell_nnz, const int coo_nnz, const int ell_max_row, 
                                                     const int nrow, const int ncol) {

  assert( ell_nnz   >= 0);
  assert( coo_nnz   >= 0);
  assert( ell_max_row >= 0);

  assert( ncol  >= 0);
  assert( nrow  >= 0);
  
  if (this->get_nnz() > 0)
    this->Clear();

  if (ell_nnz + coo_nnz > 0) {
    
    // ELL
    assert(ell_nnz == ell_max_row*nrow);

    allocate_gpu(ell_nnz, &this->mat_.ELL.val);
    allocate_gpu(ell_nnz, &this->mat_.ELL.col);
    
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    ell_nnz, this->mat_.ELL.val);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    ell_nnz, this->mat_.ELL.col);

    this->mat_.ELL.max_row = ell_max_row;
    this->ell_nnz_ = ell_nnz;

    // COO
    allocate_gpu(coo_nnz, &this->mat_.COO.row);
    allocate_gpu(coo_nnz, &this->mat_.COO.col);
    allocate_gpu(coo_nnz, &this->mat_.COO.val);
 
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    coo_nnz, this->mat_.COO.row);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    coo_nnz, this->mat_.COO.col);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    coo_nnz, this->mat_.COO.val);
    this->coo_nnz_ = coo_nnz;

    this->nrow_ = nrow;
    this->ncol_ = ncol;
    this->nnz_  = ell_nnz + coo_nnz;

  }

}


template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::Clear() {

  if (this->get_nnz() > 0) {
    
    free_gpu(&this->mat_.COO.row);
    free_gpu(&this->mat_.COO.col);
    free_gpu(&this->mat_.COO.val);
    
    free_gpu(&this->mat_.ELL.val);
    free_gpu(&this->mat_.ELL.col);
    
    this->ell_nnz_ = 0;
    this->coo_nnz_ = 0;
    this->mat_.ELL.max_row = 0;
    
    this->nrow_ = 0;
    this->ncol_ = 0;
    this->nnz_  = 0;
    
  }
  

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixHYB<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixHYB<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateHYB(cast_mat->get_ell_nnz(), cast_mat->get_coo_nnz(), cast_mat->get_ell_max_row(),
                      cast_mat->get_nrow(), cast_mat->get_ncol());

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_ell_nnz() > 0) {

      // ELL
      hipMemcpy(this->mat_.ELL.col,     // dst
                 cast_mat->mat_.ELL.col, // src
                 this->get_ell_nnz()*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.ELL.val,     // dst
                 cast_mat->mat_.ELL.val, // src
                 this->get_ell_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

    if (this->get_coo_nnz() > 0) {

      // COO
      hipMemcpy(this->mat_.COO.row,     // dst
                 cast_mat->mat_.COO.row, // src
                 (this->get_coo_nnz())*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.COO.col,     // dst
                 cast_mat->mat_.COO.col, // src
                 this->get_coo_nnz()*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.COO.val,     // dst
                 cast_mat->mat_.COO.val, // src
                 this->get_coo_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixHYB<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixHYB<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateHYB(this->get_ell_nnz(), this->get_coo_nnz(), this->get_ell_max_row(),
                      this->get_nrow(), this->get_ncol());

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_ell_nnz() > 0) {
      
      // ELL
      hipMemcpy(cast_mat->mat_.ELL.col, // dst
                 this->mat_.ELL.col,     // src
                 this->get_ell_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(cast_mat->mat_.ELL.val, // dst
                 this->mat_.ELL.val,     // src
                 this->get_ell_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }


    if (this->get_coo_nnz() > 0) {

      // COO
      hipMemcpy(cast_mat->mat_.COO.row, // dst
                 this->mat_.COO.row,     // src
                 this->get_coo_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(cast_mat->mat_.COO.col, // dst
                 this->mat_.COO.col,     // src
                 this->get_coo_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(cast_mat->mat_.COO.val, // dst
                 this->mat_.COO.val,     // src
                 this->get_coo_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
    }

  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixHYB<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixHYB<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateHYB(gpu_cast_mat->get_ell_nnz(), gpu_cast_mat->get_coo_nnz(), gpu_cast_mat->get_ell_max_row(),
                      gpu_cast_mat->get_nrow(), gpu_cast_mat->get_ncol());

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());


    if (this->get_ell_nnz() > 0) {

      // ELL
      hipMemcpy(this->mat_.ELL.col,     // dst
                 gpu_cast_mat->mat_.ELL.col, // src
                 this->get_ell_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.ELL.val,     // dst
                 gpu_cast_mat->mat_.ELL.val, // src
                 this->get_ell_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

    if (this->get_coo_nnz() > 0) {

      // COO
      hipMemcpy(this->mat_.COO.row,     // dst
                 gpu_cast_mat->mat_.COO.row, // src
                 (this->get_coo_nnz())*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.COO.col,     // dst
                 gpu_cast_mat->mat_.COO.col, // src
                 this->get_coo_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.COO.val,     // dst
                 gpu_cast_mat->mat_.COO.val, // src
                 this->get_coo_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    }
      
  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixHYB<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if (gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixHYB<ValueType>*> (dst)) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateHYB(this->get_ell_nnz(), this->get_coo_nnz(), this->get_ell_max_row(),
                      this->get_nrow(), this->get_ncol());

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_ell_nnz() > 0) {

      // ELL
      hipMemcpy(gpu_cast_mat->mat_.ELL.col, // dst
                 this->mat_.ELL.col,     // src
                 this->get_ell_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.ELL.val, // dst
                 this->mat_.ELL.val,     // src
                 this->get_ell_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

    if (this->get_coo_nnz() > 0) {

      // COO
      hipMemcpy(gpu_cast_mat->mat_.COO.row, // dst
                 this->mat_.COO.row,     // src
                 this->get_coo_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.COO.col, // dst
                 this->mat_.COO.col,     // src
                 this->get_coo_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.COO.val, // dst
                 this->mat_.COO.val,     // src
                 this->get_coo_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    }
   
    
  } else {

    //GPU to CPU
    if (host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::CopyFromHostAsync(const HostMatrix<ValueType> &src) {

  const HostMatrixHYB<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixHYB<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateHYB(cast_mat->get_ell_nnz(), cast_mat->get_coo_nnz(), cast_mat->get_ell_max_row(),
                      cast_mat->get_nrow(), cast_mat->get_ncol());

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_ell_nnz() > 0) {

      // ELL
      hipMemcpyAsync(this->mat_.ELL.col,     // dst
                      cast_mat->mat_.ELL.col, // src
                      this->get_ell_nnz()*sizeof(int), // size
                      hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(this->mat_.ELL.val,     // dst
                      cast_mat->mat_.ELL.val, // src
                      this->get_ell_nnz()*sizeof(ValueType), // size
                      hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

    if (this->get_coo_nnz() > 0) {

      // COO
      hipMemcpyAsync(this->mat_.COO.row,     // dst
                      cast_mat->mat_.COO.row, // src
                      (this->get_coo_nnz())*sizeof(int), // size
                      hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(this->mat_.COO.col,     // dst
                      cast_mat->mat_.COO.col, // src
                      this->get_coo_nnz()*sizeof(int), // size
                      hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(this->mat_.COO.val,     // dst
                      cast_mat->mat_.COO.val, // src
                      this->get_coo_nnz()*sizeof(ValueType), // size
                      hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::CopyToHostAsync(HostMatrix<ValueType> *dst) const {

  HostMatrixHYB<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixHYB<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateHYB(this->get_ell_nnz(), this->get_coo_nnz(), this->get_ell_max_row(),
                      this->get_nrow(), this->get_ncol());

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_ell_nnz() > 0) {
      
      // ELL
      hipMemcpyAsync(cast_mat->mat_.ELL.col, // dst
                      this->mat_.ELL.col,     // src
                      this->get_ell_nnz()*sizeof(int), // size
                      hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(cast_mat->mat_.ELL.val, // dst
                      this->mat_.ELL.val,     // src
                      this->get_ell_nnz()*sizeof(ValueType), // size
                      hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }


    if (this->get_coo_nnz() > 0) {

      // COO
      hipMemcpyAsync(cast_mat->mat_.COO.row, // dst
                      this->mat_.COO.row,     // src
                      this->get_coo_nnz()*sizeof(int), // size
                      hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(cast_mat->mat_.COO.col, // dst
                      this->mat_.COO.col,     // src
                      this->get_coo_nnz()*sizeof(int), // size
                      hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(cast_mat->mat_.COO.val, // dst
                      this->mat_.COO.val,     // src
                      this->get_coo_nnz()*sizeof(ValueType), // size
                      hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
    }

  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::CopyFromAsync(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixHYB<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixHYB<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateHYB(gpu_cast_mat->get_ell_nnz(), gpu_cast_mat->get_coo_nnz(), gpu_cast_mat->get_ell_max_row(),
                      gpu_cast_mat->get_nrow(), gpu_cast_mat->get_ncol());

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());


    if (this->get_ell_nnz() > 0) {

      // ELL
      hipMemcpy(this->mat_.ELL.col,     // dst
                 gpu_cast_mat->mat_.ELL.col, // src
                 this->get_ell_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.ELL.val,     // dst
                 gpu_cast_mat->mat_.ELL.val, // src
                 this->get_ell_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

    if (this->get_coo_nnz() > 0) {

      // COO
      hipMemcpy(this->mat_.COO.row,     // dst
                 gpu_cast_mat->mat_.COO.row, // src
                 (this->get_coo_nnz())*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.COO.col,     // dst
                 gpu_cast_mat->mat_.COO.col, // src
                 this->get_coo_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.COO.val,     // dst
                 gpu_cast_mat->mat_.COO.val, // src
                 this->get_coo_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    }
      
  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHostAsync(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::CopyToAsync(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixHYB<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if (gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixHYB<ValueType>*> (dst)) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateHYB(this->get_ell_nnz(), this->get_coo_nnz(), this->get_ell_max_row(),
                      this->get_nrow(), this->get_ncol());

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_ell_nnz() > 0) {

      // ELL
      hipMemcpy(gpu_cast_mat->mat_.ELL.col, // dst
                 this->mat_.ELL.col,     // src
                 this->get_ell_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.ELL.val, // dst
                 this->mat_.ELL.val,     // src
                 this->get_ell_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

    if (this->get_coo_nnz() > 0) {

      // COO
      hipMemcpy(gpu_cast_mat->mat_.COO.row, // dst
                 this->mat_.COO.row,     // src
                 this->get_coo_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.COO.col, // dst
                 this->mat_.COO.col,     // src
                 this->get_coo_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.COO.val, // dst
                 this->mat_.COO.val,     // src
                 this->get_coo_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     

    }
   
    
  } else {

    //GPU to CPU
    if (host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) {
      
      this->CopyToHostAsync(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}


template <typename ValueType>
bool GPUAcceleratorMatrixHYB<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;

  const GPUAcceleratorMatrixHYB<ValueType>   *cast_mat_hyb;
  
  if ((cast_mat_hyb = dynamic_cast<const GPUAcceleratorMatrixHYB<ValueType>*> (&mat)) != NULL) {

    this->CopyFrom(*cast_mat_hyb);
    return true;

  }

  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {

    this->Clear();

    int nrow = cast_mat_csr->get_nrow();
    int ncol = cast_mat_csr->get_ncol();
    int max_row = cast_mat_csr->get_nnz() / nrow;

    // get nnz per row for COO part
    int *nnz_coo = NULL;
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    allocate_gpu<int>(nrow, &nnz_coo);

    kernel_ell_nnz_coo<int> <<<GridSize, BlockSize>>> (nrow, max_row,
                                                       cast_mat_csr->mat_.row_offset,
                                                       nnz_coo);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    // get nnz for COO part by summing up nnz per row array
    int *d_buffer = NULL;
    int *h_buffer = NULL;
    int GROUP_SIZE;
    int LOCAL_SIZE;
    int FinalReduceSize;

    allocate_gpu<int>(this->local_backend_.GPU_warp * 4, &d_buffer);

    dim3 GridSize2(this->local_backend_.GPU_warp * 4);

    GROUP_SIZE = ( size_t( ( size_t( nrow / ( this->local_backend_.GPU_warp * 4 ) ) + 1 ) 
                 / this->local_backend_.GPU_block_size ) + 1 ) * this->local_backend_.GPU_block_size;
    LOCAL_SIZE = GROUP_SIZE / this->local_backend_.GPU_block_size;

    kernel_reduce<int, int, 256> <<<GridSize2, BlockSize>>> (nrow, nnz_coo, d_buffer, GROUP_SIZE, LOCAL_SIZE);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    FinalReduceSize = this->local_backend_.GPU_warp * 4;
    allocate_host(FinalReduceSize, &h_buffer);

    hipMemcpy(h_buffer, // dst
               d_buffer, // src
               FinalReduceSize*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    free_gpu<int>(&d_buffer);

    int num_nnz_coo = 0;
    for ( int i=0; i<FinalReduceSize; ++i )
      num_nnz_coo += h_buffer[i];

    free_host(&h_buffer);

    // allocate ELL and COO matrices
    int num_nnz_ell = max_row * nrow;

    assert(num_nnz_ell > 0);
    assert(num_nnz_coo > 0);

    this->AllocateHYB(num_nnz_ell, num_nnz_coo, max_row, nrow, ncol);

    hipMemset(this->mat_.ELL.col, -1, num_nnz_ell*sizeof(int));
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    // copy up to num_cols_per_row values of row i into the ELL
    int *nnz_ell = NULL;

    allocate_gpu<int>(nrow, &nnz_ell);

    kernel_ell_fill_ell<ValueType, int> <<<GridSize, BlockSize>>> (nrow, max_row,
                                                                   cast_mat_csr->mat_.row_offset,
                                                                   cast_mat_csr->mat_.col,
                                                                   cast_mat_csr->mat_.val,
                                                                   this->mat_.ELL.col,
                                                                   this->mat_.ELL.val,
                                                                   nnz_ell);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    // TODO currently performing partial sum on host
    allocate_host(nrow, &h_buffer);
    hipMemcpy(h_buffer, // dst
               nnz_ell, // src
               nrow*sizeof(int), // size
               hipMemcpyDeviceToHost);

    for (int i=1; i<nrow; ++i)
      h_buffer[i] += h_buffer[i-1];

    hipMemcpy(nnz_ell, // dst
               h_buffer, // src
               nrow*sizeof(int), // size
               hipMemcpyHostToDevice);

    free_host(&h_buffer);
    // end TODO

    // copy any remaining values in row i into the COO

    kernel_ell_fill_coo<ValueType, int> <<<GridSize, BlockSize>>> (nrow, cast_mat_csr->mat_.row_offset,
                                                                   cast_mat_csr->mat_.col,
                                                                   cast_mat_csr->mat_.val,
                                                                   nnz_coo, nnz_ell,
                                                                   this->mat_.COO.row, this->mat_.COO.col,
                                                                   this->mat_.COO.val);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    free_gpu<int>(&nnz_ell);
    free_gpu<int>(&nnz_coo);

    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = num_nnz_ell + num_nnz_coo;
    this->mat_.ELL.max_row = max_row;
    this->ell_nnz_ = num_nnz_ell;
    this->coo_nnz_ = num_nnz_coo;

    return true;

  }

  return false;

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::Apply(const BaseVector<ValueType> &in, BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {
    
    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());
    
    
    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in) ; 
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out) ; 
    
    assert(cast_in != NULL);
    assert(cast_out!= NULL);


    // ELL
    if (this->get_ell_nnz() > 0) {

      int nrow = this->get_nrow();
      int ncol = this->get_ncol();
      int max_row = this->get_ell_max_row();
      dim3 BlockSize(this->local_backend_.GPU_block_size);
      dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);
    
      
      kernel_ell_spmv<ValueType, int> <<<GridSize, BlockSize>>> (nrow, ncol, max_row,
                                                                 this->mat_.ELL.col, this->mat_.ELL.val,
                                                                 cast_in->vec_, cast_out->vec_ );
      
      CHECK_CUDA_ERROR(__FILE__, __LINE__);
    }


    if (this->get_coo_nnz() > 0) {

      // do not support super small matrices
      assert(this->get_coo_nnz() > this->local_backend_.GPU_warp); 
      
      // ----------------------------------------------------------
      // Modified and adapted from CUSP 0.3.1, 
      // http://code.google.com/p/cusp-library/
      // NVIDIA, APACHE LICENSE 2.0
      // ----------------------------------------------------------
      // see __spmv_coo_flat(...)
      // ----------------------------------------------------------
      // CHANGELOG
      // - adapted interface
      // ----------------------------------------------------------  


      const unsigned int BLOCK_SIZE = this->local_backend_.GPU_block_size;
      //    const unsigned int MAX_BLOCKS = this->local_backend_.GPU_max_blocks;
      
      const unsigned int MAX_BLOCKS = 32; //  cusp::detail::device::arch::max_active_blocks(spmv_coo_flat_kernel<IndexType, ValueType, BLOCK_SIZE, UseCache>, BLOCK_SIZE, (size_t) 0);
      
      const unsigned int WARPS_PER_BLOCK = BLOCK_SIZE / this->local_backend_.GPU_warp;
      
      
      const unsigned int num_units  = this->get_coo_nnz() / this->local_backend_.GPU_warp; 
      const unsigned int num_warps  = std::min(num_units, WARPS_PER_BLOCK * MAX_BLOCKS);
      const unsigned int num_blocks = (num_warps + (WARPS_PER_BLOCK-1)) / WARPS_PER_BLOCK; // (N + (granularity - 1)) / granularity
      const unsigned int num_iters  = (num_units +  (num_warps-1)) / num_warps;
      
      const unsigned int interval_size = this->local_backend_.GPU_warp * num_iters;
      
      const int tail = num_units * this->local_backend_.GPU_warp; // do the last few nonzeros separately (fewer than this->local_backend_.GPU_warp elements)
      
      const unsigned int active_warps = (interval_size == 0) ? 0 : ((tail + (interval_size-1))/interval_size);
      
      int *temp_rows = NULL;
      ValueType *temp_vals = NULL;
      
      allocate_gpu(active_warps, &temp_rows);
      allocate_gpu(active_warps, &temp_vals);
      
      
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      // TODO 
      // BLOCK_SIZE == 256 
      // WARP_SIZE == 32
      kernel_spmv_coo_flat<int, ValueType, 256, 32> <<<num_blocks, BLOCK_SIZE>>>
        (tail, interval_size, 
         this->mat_.COO.row, this->mat_.COO.col, this->mat_.COO.val, 
         ValueType(1.0),
         cast_in->vec_, cast_out->vec_,
         temp_rows, temp_vals);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      // TODO 
      // BLOCK_SIZE == 256 
      kernel_spmv_coo_reduce_update<int, ValueType, 256> <<<1, BLOCK_SIZE>>>
        (active_warps, temp_rows, temp_vals, cast_out->vec_);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      kernel_spmv_coo_serial<int, ValueType> <<<1,1>>>
        (this->get_coo_nnz() - tail, 
         this->mat_.COO.row + tail, 
         this->mat_.COO.col + tail, 
         this->mat_.COO.val + tail, 
         ValueType(1.0),
         cast_in->vec_, cast_out->vec_);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      free_gpu(&temp_rows);
      free_gpu(&temp_vals);

    }

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixHYB<ValueType>::ApplyAdd(const BaseVector<ValueType> &in, const ValueType scalar,
                                                  BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {
    
    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());
    
    
    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in) ; 
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out) ; 
    
    assert(cast_in != NULL);
    assert(cast_out!= NULL);


    // ELL
    if (this->get_ell_nnz() > 0) {

      int nrow = this->get_nrow();
      int ncol = this->get_ncol();
      int max_row = this->get_ell_max_row();
      dim3 BlockSize(this->local_backend_.GPU_block_size);
      dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);
    
      
      kernel_ell_add_spmv<ValueType, int> <<<GridSize, BlockSize>>> (nrow, ncol, max_row,
                                                                     this->mat_.ELL.col, this->mat_.ELL.val,
                                                                     scalar,
                                                                     cast_in->vec_, cast_out->vec_ );
      
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }


    if (this->get_coo_nnz() > 0) {

      // do not support super small matrices
      assert(this->get_coo_nnz() > this->local_backend_.GPU_warp); 
      
      // ----------------------------------------------------------
      // Modified and adapted from CUSP 0.3.1, 
      // http://code.google.com/p/cusp-library/
      // NVIDIA, APACHE LICENSE 2.0
      // ----------------------------------------------------------
      // see __spmv_coo_flat(...)
      // ----------------------------------------------------------
      // CHANGELOG
      // - adapted interface
      // ----------------------------------------------------------  


      const unsigned int BLOCK_SIZE = this->local_backend_.GPU_block_size;
      //    const unsigned int MAX_BLOCKS = this->local_backend_.GPU_max_blocks;
      
      const unsigned int MAX_BLOCKS = 32; //  cusp::detail::device::arch::max_active_blocks(spmv_coo_flat_kernel<IndexType, ValueType, BLOCK_SIZE, UseCache>, BLOCK_SIZE, (size_t) 0);
      
      const unsigned int WARPS_PER_BLOCK = BLOCK_SIZE / this->local_backend_.GPU_warp;
      
      
      const unsigned int num_units  = this->get_coo_nnz() / this->local_backend_.GPU_warp; 
      const unsigned int num_warps  = std::min(num_units, WARPS_PER_BLOCK * MAX_BLOCKS);
      const unsigned int num_blocks = (num_warps + (WARPS_PER_BLOCK-1)) / WARPS_PER_BLOCK; // (N + (granularity - 1)) / granularity
      const unsigned int num_iters  = (num_units +  (num_warps-1)) / num_warps;
      
      const unsigned int interval_size = this->local_backend_.GPU_warp * num_iters;
      
      const int tail = num_units * this->local_backend_.GPU_warp; // do the last few nonzeros separately (fewer than this->local_backend_.GPU_warp elements)
      
      const unsigned int active_warps = (interval_size == 0) ? 0 : ((tail + (interval_size-1))/interval_size);
      
      int *temp_rows = NULL;
      ValueType *temp_vals = NULL;
      
      allocate_gpu(active_warps, &temp_rows);
      allocate_gpu(active_warps, &temp_vals);
      
      
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      // TODO 
      // BLOCK_SIZE == 256 
      // WARP_SIZE == 32
      kernel_spmv_coo_flat<int, ValueType, 256, 32> <<<num_blocks, BLOCK_SIZE>>>
        (tail, interval_size, 
         this->mat_.COO.row, this->mat_.COO.col, this->mat_.COO.val, 
         scalar,
         cast_in->vec_, cast_out->vec_,
         temp_rows, temp_vals);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      // TODO 
      // BLOCK_SIZE == 256 
      kernel_spmv_coo_reduce_update<int, ValueType, 256> <<<1, BLOCK_SIZE>>>
        (active_warps, temp_rows, temp_vals, cast_out->vec_);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      kernel_spmv_coo_serial<int, ValueType> <<<1,1>>>
        (this->get_coo_nnz() - tail, 
         this->mat_.COO.row + tail, 
         this->mat_.COO.col + tail, 
         this->mat_.COO.val + tail, 
         scalar,
         cast_in->vec_, cast_out->vec_);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      free_gpu(&temp_rows);
      free_gpu(&temp_vals);
      

    }

  }

}


template class GPUAcceleratorMatrixHYB<double>;
template class GPUAcceleratorMatrixHYB<float>;

}
