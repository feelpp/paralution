#include "hip/hip_runtime.h"
// **************************************************************************
//
//    PARALUTION   www.paralution.com
//
//    Copyright (C) 2015  PARALUTION Labs UG (haftungsbeschränkt) & Co. KG
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRA 706051
//                        Vertreten durch:
//                        PARALUTION Labs Verwaltungs UG (haftungsbeschränkt)
//                        Am Hasensprung 6, 76571 Gaggenau
//                        Handelsregister: Amtsgericht Mannheim, HRB 721277
//                        Geschäftsführer: Dimitar Lukarski, Nico Trost
//
//    This program is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    This program is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
// **************************************************************************



// PARALUTION version 1.0.0 


#include "../../utils/def.hpp"
#include "gpu_matrix_csr.hpp"
#include "gpu_matrix_ell.hpp"
#include "gpu_vector.hpp"
#include "../host/host_matrix_ell.hpp"
#include "../base_matrix.hpp"
#include "../base_vector.hpp"
#include "../backend_manager.hpp"
#include "../../utils/log.hpp"
#include "gpu_utils.hpp"
#include "cuda_kernels_general.hpp"
#include "cuda_kernels_ell.hpp"
#include "gpu_allocate_free.hpp"
#include "../../utils/allocate_free.hpp"
#include "../matrix_formats_ind.hpp"

#include <hip/hip_runtime.h>

namespace paralution {

template <typename ValueType>
GPUAcceleratorMatrixELL<ValueType>::GPUAcceleratorMatrixELL() {

  // no default constructors
  LOG_INFO("no default constructor");
  FATAL_ERROR(__FILE__, __LINE__);

}

template <typename ValueType>
GPUAcceleratorMatrixELL<ValueType>::GPUAcceleratorMatrixELL(const Paralution_Backend_Descriptor local_backend) {

  LOG_DEBUG(this, "GPUAcceleratorMatrixELL::GPUAcceleratorMatrixELL()",
            "constructor with local_backend");

  this->mat_.val = NULL;
  this->mat_.col = NULL;
  this->mat_.max_row = 0;
  this->set_backend(local_backend); 

  CHECK_CUDA_ERROR(__FILE__, __LINE__);

}


template <typename ValueType>
GPUAcceleratorMatrixELL<ValueType>::~GPUAcceleratorMatrixELL() {

  LOG_DEBUG(this, "GPUAcceleratorMatrixELL::~GPUAcceleratorMatrixELL()",
            "destructor");

  this->Clear();

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::info(void) const {

  LOG_INFO("GPUAcceleratorMatrixELL<ValueType>");

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::AllocateELL(const int nnz, const int nrow, const int ncol, const int max_row) {

  assert( nnz   >= 0);
  assert( ncol  >= 0);
  assert( nrow  >= 0);
  assert( max_row >= 0);

  if (this->get_nnz() > 0)
    this->Clear();

  if (nnz > 0) {

    assert(nnz == max_row * nrow);

    allocate_gpu(nnz, &this->mat_.val);
    allocate_gpu(nnz, &this->mat_.col);
    
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, this->mat_.val);
    set_to_zero_gpu(this->local_backend_.GPU_block_size, 
                    this->local_backend_.GPU_max_threads,
                    nnz, this->mat_.col);
    
    this->mat_.max_row = max_row;
    this->nrow_ = nrow;
    this->ncol_ = ncol;
    this->nnz_  = nnz;

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::Clear() {

  if (this->get_nnz() > 0) {

    free_gpu(&this->mat_.val);
    free_gpu(&this->mat_.col);

    this->nrow_ = 0;
    this->ncol_ = 0;
    this->nnz_  = 0;

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::SetDataPtrELL(int **col, ValueType **val,
                                             const int nnz, const int nrow, const int ncol, const int max_row) {

  assert(*col != NULL);
  assert(*val != NULL);
  assert(nnz > 0);
  assert(nrow > 0);
  assert(ncol > 0);
  assert(max_row > 0);
  assert(max_row*nrow == nnz);

  this->Clear();

  hipDeviceSynchronize();

  this->mat_.max_row = max_row;
  this->nrow_ = nrow;
  this->ncol_ = ncol;
  this->nnz_  = nnz;

  this->mat_.col = *col;
  this->mat_.val = *val;

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::LeaveDataPtrELL(int **col, ValueType **val, int &max_row) {

  assert(this->nrow_ > 0);
  assert(this->ncol_ > 0);
  assert(this->nnz_ > 0);
  assert(this->mat_.max_row > 0);
  assert(this->mat_.max_row*this->nrow_ == this->nnz_);

  hipDeviceSynchronize();

  // see free_host function for details
  *col = this->mat_.col;
  *val = this->mat_.val;

  this->mat_.col = NULL;
  this->mat_.val = NULL;

  max_row = this->mat_.max_row;

  this->mat_.max_row = 0;
  this->nrow_ = 0;
  this->ncol_ = 0;
  this->nnz_  = 0;

}




template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::CopyFromHost(const HostMatrix<ValueType> &src) {

  const HostMatrixELL<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixELL<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateELL(cast_mat->get_nnz(), cast_mat->get_nrow(), cast_mat->get_ncol(), cast_mat->get_max_row());

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) { 

      hipMemcpy(this->mat_.col,     // dst
                 cast_mat->mat_.col, // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.val,     // dst
                 cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::CopyToHost(HostMatrix<ValueType> *dst) const {

  HostMatrixELL<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixELL<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateELL(this->get_nnz(), this->get_nrow(), this->get_ncol(), this->get_max_row() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpy(cast_mat->mat_.col, // dst
                 this->mat_.col,     // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(cast_mat->mat_.val, // dst
                 this->mat_.val,     // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::CopyFrom(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixELL<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixELL<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateELL(gpu_cast_mat->get_nnz(), gpu_cast_mat->get_nrow(), gpu_cast_mat->get_ncol(), gpu_cast_mat->get_max_row() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.col,         // dst
                 gpu_cast_mat->mat_.col, // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.val,         // dst
                 gpu_cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHost(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::CopyTo(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixELL<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixELL<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateELL(gpu_cast_mat->get_nnz(), gpu_cast_mat->get_nrow(), gpu_cast_mat->get_ncol(), gpu_cast_mat->get_max_row() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {
      
      hipMemcpy(gpu_cast_mat->mat_.col, // dst
                 this->mat_.col,         // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.val, // dst
                 this->mat_.val,         // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHost(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::CopyFromHostAsync(const HostMatrix<ValueType> &src) {

  const HostMatrixELL<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // CPU to GPU copy
  if ((cast_mat = dynamic_cast<const HostMatrixELL<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateELL(cast_mat->get_nnz(), cast_mat->get_nrow(), cast_mat->get_ncol(), cast_mat->get_max_row());

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) { 

      hipMemcpyAsync(this->mat_.col,     // dst
                      cast_mat->mat_.col, // src
                      this->get_nnz()*sizeof(int), // size
                      hipMemcpyHostToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(this->mat_.val,     // dst
                      cast_mat->mat_.val, // src
                      this->get_nnz()*sizeof(ValueType), // size
                      hipMemcpyHostToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    src.info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::CopyToHostAsync(HostMatrix<ValueType> *dst) const {

  HostMatrixELL<ValueType> *cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to CPU copy
  if ((cast_mat = dynamic_cast<HostMatrixELL<ValueType>*> (dst)) != NULL) {

    cast_mat->set_backend(this->local_backend_);   

  if (dst->get_nnz() == 0)
    cast_mat->AllocateELL(this->get_nnz(), this->get_nrow(), this->get_ncol(), this->get_max_row() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpyAsync(cast_mat->mat_.col, // dst
                      this->mat_.col,     // src
                      this->get_nnz()*sizeof(int), // size
                      hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpyAsync(cast_mat->mat_.val, // dst
                      this->mat_.val,     // src
                      this->get_nnz()*sizeof(ValueType), // size
                      hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {
    
    LOG_INFO("Error unsupported GPU matrix type");
    this->info();
    dst->info();
    FATAL_ERROR(__FILE__, __LINE__);
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::CopyFromAsync(const BaseMatrix<ValueType> &src) {

  const GPUAcceleratorMatrixELL<ValueType> *gpu_cast_mat;
  const HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == src.get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<const GPUAcceleratorMatrixELL<ValueType>*> (&src)) != NULL) {
    
  if (this->get_nnz() == 0)
    this->AllocateELL(gpu_cast_mat->get_nnz(), gpu_cast_mat->get_nrow(), gpu_cast_mat->get_ncol(), gpu_cast_mat->get_max_row() );

    assert(this->get_nnz()  == src.get_nnz());
    assert(this->get_nrow() == src.get_nrow());
    assert(this->get_ncol() == src.get_ncol());

    if (this->get_nnz() > 0) {

      hipMemcpy(this->mat_.col,         // dst
                 gpu_cast_mat->mat_.col, // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToDevice);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(this->mat_.val,         // dst
                 gpu_cast_mat->mat_.val, // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToDevice);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }

  } else {

    //CPU to GPU
    if ((host_cast_mat = dynamic_cast<const HostMatrix<ValueType>*> (&src)) != NULL) {
      
      this->CopyFromHostAsync(*host_cast_mat);
      
    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      src.info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }
    
  }

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::CopyToAsync(BaseMatrix<ValueType> *dst) const {

  GPUAcceleratorMatrixELL<ValueType> *gpu_cast_mat;
  HostMatrix<ValueType> *host_cast_mat;

  // copy only in the same format
  assert(this->get_mat_format() == dst->get_mat_format());

  // GPU to GPU copy
  if ((gpu_cast_mat = dynamic_cast<GPUAcceleratorMatrixELL<ValueType>*> (dst)) != NULL) {

    gpu_cast_mat->set_backend(this->local_backend_);       

  if (this->get_nnz() == 0)
    gpu_cast_mat->AllocateELL(gpu_cast_mat->get_nnz(), gpu_cast_mat->get_nrow(), gpu_cast_mat->get_ncol(), gpu_cast_mat->get_max_row() );

    assert(this->get_nnz()  == dst->get_nnz());
    assert(this->get_nrow() == dst->get_nrow());
    assert(this->get_ncol() == dst->get_ncol());

    if (this->get_nnz() > 0) {
      
      hipMemcpy(gpu_cast_mat->mat_.col, // dst
                 this->mat_.col,         // src
                 this->get_nnz()*sizeof(int), // size
                 hipMemcpyDeviceToHost);
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
      
      hipMemcpy(gpu_cast_mat->mat_.val, // dst
                 this->mat_.val,         // src
                 this->get_nnz()*sizeof(ValueType), // size
                 hipMemcpyDeviceToHost);    
      CHECK_CUDA_ERROR(__FILE__, __LINE__);     
    }
    
  } else {

    //GPU to CPU
    if ((host_cast_mat = dynamic_cast<HostMatrix<ValueType>*> (dst)) != NULL) {
      
      this->CopyToHostAsync(host_cast_mat);

    } else {
      
      LOG_INFO("Error unsupported GPU matrix type");
      this->info();
      dst->info();
      FATAL_ERROR(__FILE__, __LINE__);
      
    }

  }


}

template <typename ValueType>
bool GPUAcceleratorMatrixELL<ValueType>::ConvertFrom(const BaseMatrix<ValueType> &mat) {

  this->Clear();

  // empty matrix is empty matrix
  if (mat.get_nnz() == 0)
    return true;

  const GPUAcceleratorMatrixELL<ValueType> *cast_mat_ell;

  if ((cast_mat_ell = dynamic_cast<const GPUAcceleratorMatrixELL<ValueType>*> (&mat)) != NULL) {

    this->CopyFrom(*cast_mat_ell);
    return true;

  }

  const GPUAcceleratorMatrixCSR<ValueType>   *cast_mat_csr;
  if ((cast_mat_csr = dynamic_cast<const GPUAcceleratorMatrixCSR<ValueType>*> (&mat)) != NULL) {

    this->Clear();

    assert(cast_mat_csr->get_nrow() > 0);
    assert(cast_mat_csr->get_ncol() > 0);
    assert(cast_mat_csr->get_nnz() > 0);

    int max_row = 0;
    int nrow = cast_mat_csr->get_nrow();

    int *d_buffer = NULL;
    int *h_buffer = NULL;
    int GROUP_SIZE;
    int LOCAL_SIZE;
    int FinalReduceSize;

    allocate_gpu<int>(this->local_backend_.GPU_warp * 4, &d_buffer);

    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(this->local_backend_.GPU_warp * 4);

    GROUP_SIZE = ( size_t( ( size_t( nrow / ( this->local_backend_.GPU_warp * 4 ) ) + 1 ) 
                 / this->local_backend_.GPU_block_size ) + 1 ) * this->local_backend_.GPU_block_size;
    LOCAL_SIZE = GROUP_SIZE / this->local_backend_.GPU_block_size;

    kernel_ell_max_row<int, int, 256> <<<GridSize, BlockSize>>> (nrow, cast_mat_csr->mat_.row_offset,
                                                                 d_buffer, GROUP_SIZE, LOCAL_SIZE);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    FinalReduceSize = this->local_backend_.GPU_warp * 4;
    allocate_host(FinalReduceSize, &h_buffer);

    hipMemcpy(h_buffer, // dst
               d_buffer, // src
               FinalReduceSize*sizeof(int), // size
               hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    free_gpu<int>(&d_buffer);

    for ( int i=0; i<FinalReduceSize; ++i )
      if (max_row < h_buffer[i]) max_row = h_buffer[i];

    free_host(&h_buffer);

    int nnz_ell = max_row * nrow;

    this->AllocateELL(nnz_ell, nrow, cast_mat_csr->get_ncol(), max_row);

    set_to_zero_gpu(this->local_backend_.GPU_block_size,
                    this->local_backend_.GPU_max_threads,
                    nnz_ell, this->mat_.val);

    set_to_zero_gpu(this->local_backend_.GPU_block_size,
                    this->local_backend_.GPU_max_threads,
                    nnz_ell, this->mat_.col);

    dim3 BlockSize2(this->local_backend_.GPU_block_size);
    dim3 GridSize2(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_ell_csr_to_ell<ValueType, int> <<<GridSize2, BlockSize2>>> (nrow, max_row, cast_mat_csr->mat_.row_offset,
                                                                       cast_mat_csr->mat_.col, cast_mat_csr->mat_.val,
                                                                       this->mat_.col, this->mat_.val);
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

    this->mat_.max_row = max_row;
    this->nrow_ = cast_mat_csr->get_nrow();
    this->ncol_ = cast_mat_csr->get_ncol();
    this->nnz_  = max_row * nrow;

    return true;

  }

  return false;

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::Apply(const BaseVector<ValueType> &in, BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in);
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    int nrow = this->get_nrow();
    int ncol = this->get_ncol();
    int max_row = this->get_max_row();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_ell_spmv<ValueType, int> <<<GridSize, BlockSize>>> (nrow, ncol, max_row,
                                                               this->mat_.col, this->mat_.val,
                                                               cast_in->vec_, cast_out->vec_ );
    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}

template <typename ValueType>
void GPUAcceleratorMatrixELL<ValueType>::ApplyAdd(const BaseVector<ValueType> &in, const ValueType scalar,
                                                  BaseVector<ValueType> *out) const {

  if (this->get_nnz() > 0) {

    assert(in.  get_size() >= 0);
    assert(out->get_size() >= 0);
    assert(in.  get_size() == this->get_ncol());
    assert(out->get_size() == this->get_nrow());

    const GPUAcceleratorVector<ValueType> *cast_in = dynamic_cast<const GPUAcceleratorVector<ValueType>*> (&in);
    GPUAcceleratorVector<ValueType> *cast_out      = dynamic_cast<      GPUAcceleratorVector<ValueType>*> (out);

    assert(cast_in != NULL);
    assert(cast_out!= NULL);

    int nrow = this->get_nrow();
    int ncol = this->get_ncol();
    int max_row = this->get_max_row();
    dim3 BlockSize(this->local_backend_.GPU_block_size);
    dim3 GridSize(nrow / this->local_backend_.GPU_block_size + 1);

    kernel_ell_add_spmv<ValueType, int> <<<GridSize, BlockSize>>> (nrow, ncol, max_row,
                                                                   this->mat_.col, this->mat_.val,
                                                                   scalar,
                                                                   cast_in->vec_, cast_out->vec_);

    CHECK_CUDA_ERROR(__FILE__, __LINE__);

  }

}


template class GPUAcceleratorMatrixELL<double>;
template class GPUAcceleratorMatrixELL<float>;

}
